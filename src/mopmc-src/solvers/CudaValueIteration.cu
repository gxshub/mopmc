#include "hip/hip_runtime.h"
//
// Created by guoxin on 15/11/23.
//

#include "CudaFunctions.h"
#include "CudaValueIteration.cuh"
#include "mopmc-src/solvers/CudaWarmUp.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <iostream>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>


#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("cuSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUBLAS(func)                                                     \
{                                                                              \
    hipblasStatus_t status = (func);                                            \
    if (status != HIPBLAS_STATUS_SUCCESS) {                                     \
        printf("CUBLAS API failed at line %d with error: %d\n",                \
               __LINE__, status);                                              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

namespace mopmc {
    namespace value_iteration {
        namespace gpu {
            template<typename ValueType>
            CudaValueIterationHandler<ValueType>::CudaValueIterationHandler(QueryData<ValueType, int> *queryData) :
                    data(queryData),
                    transitionMatrix(queryData->transitionMatrix),
                    rowGroupIndices(queryData->rowGroupIndices),
                    row2RowGroupMapping(queryData->row2RowGroupMapping),
                    flattenRewardVector(queryData->flattenRewardVector),
                    scheduler(queryData->scheduler),
                    iniRow(queryData->initialRow),
                    nobjs(queryData->objectiveCount)
            {
                A_nnz = transitionMatrix.nonZeros();
                A_ncols = transitionMatrix.cols();
                A_nrows = transitionMatrix.rows();
                B_ncols = A_ncols;
                B_nrows = B_ncols;
                Z_nrows = B_ncols;
                Z_ncols = nobjs;
                Z_ld = Z_nrows;
                results.resize(nobjs + 1);
                //some assertions
                assert(A_ncols == scheduler.size());
                assert(flattenRewardVector.size() == A_nrows * nobjs);
                assert(rowGroupIndices.size() == A_ncols + 1);
            }

            template<typename ValueType>
            int CudaValueIterationHandler<ValueType>::initialize() {
                //GPU warm up
                mopmc::kernels::launchWarmupKernel();
                std::cout << ("____ CUDA INITIALIZING ____\n");
                // hipMalloc CONSTANTS -------------------------------------------------------------
                CHECK_CUDA(hipMalloc((void **) &dA_csrOffsets, (A_nrows + 1) * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dA_columns, A_nnz * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dA_values, A_nnz * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dA_rows_backup, A_nnz * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dR, A_nrows * nobjs * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dRowGroupIndices, (A_ncols + 1) * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dRow2RowGroupMapping, A_nrows * sizeof(int)))
                // hipMalloc Variables -------------------------------------------------------------
                CHECK_CUDA(hipMalloc((void **) &dX, A_ncols * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dX1, A_ncols * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dY, A_nrows * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dScheduler, A_ncols * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dW, nobjs * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dRw, A_nrows * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dResult, (nobjs + 1) * sizeof(double)))
                // hipMalloc PHASE B-------------------------------------------------------------
                CHECK_CUDA(hipMalloc((void **) &dB_csrOffsets, (A_ncols + 1) * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dB_columns, A_nnz * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dB_values, A_nnz * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dB_rows_backup, A_nnz * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dMasking_nrows, A_nrows * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dMasking_nnz, A_nnz * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dMasking_tiled, Z_ncols * A_nrows * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dRi, B_nrows * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dRPart, Z_ncols * Z_nrows * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dZ, Z_ncols * Z_nrows * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dZ1, Z_ncols * Z_nrows * sizeof(double)))
                // hipMemcpy -------------------------------------------------------------
                CHECK_CUDA(hipMemcpy(dA_csrOffsets, transitionMatrix.outerIndexPtr(), (A_nrows + 1) * sizeof(int),
                                      hipMemcpyHostToDevice))
                CHECK_CUDA(hipMemcpy(dA_columns, transitionMatrix.innerIndexPtr(), A_nnz * sizeof(int),
                                      hipMemcpyHostToDevice))
                CHECK_CUDA(hipMemcpy(dA_values, transitionMatrix.valuePtr(), A_nnz * sizeof(double),
                                      hipMemcpyHostToDevice))
                CHECK_CUDA(hipMemcpy(dR, flattenRewardVector.data(), A_nrows * nobjs * sizeof(double),
                                      hipMemcpyHostToDevice))
                CHECK_CUDA(hipMemcpy(dRowGroupIndices, rowGroupIndices.data(), (A_ncols + 1) * sizeof(int),
                                      hipMemcpyHostToDevice))
                CHECK_CUDA(hipMemcpy(dRow2RowGroupMapping, row2RowGroupMapping.data(), A_nrows * sizeof(int),
                                      hipMemcpyHostToDevice))
                CHECK_CUDA(hipMemcpy(dScheduler, scheduler.data(), A_ncols * sizeof(int), hipMemcpyHostToDevice))
                //-------------------------------------------------------------------------
                CHECK_CUSPARSE(hipsparseCreate(&handle))
                CHECK_CUBLAS(hipblasCreate(&cublasHandle))
                // Create sparse matrices A in CSR format
                CHECK_CUSPARSE(hipsparseCreateCsr(&matA, A_nrows, A_ncols, A_nnz,
                                                 dA_csrOffsets, dA_columns, dA_values,
                                                 HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                                 HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F))
                CHECK_CUSPARSE(hipsparseCreateDnMat(&matZ, Z_nrows, Z_ncols, Z_ld, dZ, HIP_R_64F, HIPSPARSE_ORDER_COL))
                CHECK_CUSPARSE(hipsparseCreateDnMat(&matZ1, Z_nrows, Z_ncols, Z_ld, dZ1, HIP_R_64F, HIPSPARSE_ORDER_COL))
                CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, A_ncols, dX, HIP_R_64F))
                CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, A_nrows, dY, HIP_R_64F))
                CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX1, A_ncols, dX1, HIP_R_64F))
                CHECK_CUSPARSE(hipsparseCreateDnVec(&vecRw, A_nrows, dRw, HIP_R_64F))
                // allocate an external buffer if needed
                CHECK_CUSPARSE(hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                       &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                                                       HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize))
                CHECK_CUDA(hipMalloc(&dBuffer, bufferSize))
                return EXIT_SUCCESS;
            }

            template<typename ValueType>
            int CudaValueIterationHandler<ValueType>::valueIteration(const std::vector<double> &w) {
                this->valueIterationPhaseOne(w);
                this->valueIterationPhaseTwo();
                return EXIT_SUCCESS;
            }

            template<typename ValueType>
            int CudaValueIterationHandler<ValueType>::valueIterationPhaseOne(const std::vector<double> &w, bool toHost) {
                //std::cout << "____ VI PHASE ONE ____\n" ;
                CHECK_CUDA(hipMemcpy(dW, w.data(), nobjs * sizeof(double), hipMemcpyHostToDevice))
                mopmc::functions::cuda::aggregateLauncher(dW, dR, dRw, A_nrows, nobjs);

                iteration = 0;
                do {
                    // Y = R
                    CHECK_CUBLAS(hipblasDcopy_64(cublasHandle, A_nrows, dRw, 1, dY, 1))
                    if (iteration == 0) {
                        mopmc::functions::cuda::maxValueLauncher1(dY, dX, dRowGroupIndices, dScheduler, A_ncols + 1, A_nrows);
                    }
                    // Y = A.X + Y (Y = R)
                    CHECK_CUSPARSE(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                                                HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer))
                    // X1 = X
                    CHECK_CUBLAS(hipblasDcopy_64(cublasHandle, A_ncols, dX, 1, dX1, 1))
                    // X(s) = max_{a\in Act(s)} Y(s,a), scheduler(s) = argmax_{a\in Act(s)} scheduler(s)
                    mopmc::functions::cuda::maxValueLauncher1(dY, dX, dRowGroupIndices, dScheduler, A_ncols + 1, A_nrows);
                    // X1 = -1 * X+ X1
                    CHECK_CUBLAS(hipblasDaxpy_64(cublasHandle, A_ncols, &alpha2, dX, 1, dX1, 1))
                    // max |X1|
                    CHECK_CUBLAS(hipblasIdamax(cublasHandle, A_ncols, dX1, 1, &maxInd))
                    // to get maxEps, we must reduce also by one since this is FORTRAN based indexing.
                    CHECK_CUDA(hipMemcpy(&maxEps, dX1 + maxInd - 1, sizeof(double), hipMemcpyDeviceToHost))
                    maxEps = (maxEps >= 0) ? maxEps : -maxEps;
                    //maxEps = mopmc::kernels::findMaxEps(dXPrime, A_ncols, maxEps);
                    ++iteration;
                    //printf("___ VI PHASE ONE, ITERATION %i, maxEps %f\n", iteration, maxEps);
                } while (maxEps > tolerance && iteration < maxIter);

                if (iteration == maxIter) {
                    std::cout << "[warning] loop exit after reaching maximum iteration number (" << iteration <<")\n";
                }
                //std::cout << "terminated after " << iteration <<" iterations.\n";
                //copy result
                thrust::copy(thrust::device, dX + iniRow, dX + iniRow + 1, dResult + nobjs);
                if(toHost) {
                    CHECK_CUDA(hipMemcpy(scheduler.data(), dScheduler, A_ncols * sizeof(int), hipMemcpyDeviceToHost))
                }
                return EXIT_SUCCESS;
            }

            template<typename ValueType>
            int CudaValueIterationHandler<ValueType>::valueIterationPhaseTwo() {
                //std::cout << "____ VI PHASE TWO ____\n";
                // generate a DTMC transition matrix as a csr matrix
                CHECK_CUSPARSE(hipsparseXcsr2coo(handle, dA_csrOffsets, A_nnz, A_nrows, dA_rows_backup,
                                                HIPSPARSE_INDEX_BASE_ZERO))
                mopmc::functions::cuda::binaryMaskingLauncher(dA_csrOffsets,
                                                              dRowGroupIndices, dRow2RowGroupMapping,
                                                              dScheduler, dMasking_nrows, dMasking_nnz, A_nrows);
                thrust::copy_if(thrust::device, dA_values, dA_values + A_nnz,
                                dMasking_nnz, dB_values, mopmc::functions::cuda::is_not_zero<int>());
                thrust::copy_if(thrust::device, dA_columns, dA_columns + A_nnz,
                                dMasking_nnz, dB_columns, mopmc::functions::cuda::is_not_zero<int>());
                thrust::copy_if(thrust::device, dA_rows_backup, dA_rows_backup + A_nnz,
                                dMasking_nnz, dB_rows_backup, mopmc::functions::cuda::is_not_zero<int>());
                /* @param B_nnz: number of non-zero entries in the DTMC transition matrix */
                B_nnz = (int) thrust::count_if(thrust::device, dMasking_nnz, dMasking_nnz + A_nnz,
                                               mopmc::functions::cuda::is_not_zero<double>());
                mopmc::functions::cuda::row2RowGroupLauncher(dRow2RowGroupMapping, dB_rows_backup, B_nnz);
                CHECK_CUSPARSE(hipsparseXcoo2csr(handle, dB_rows_backup, B_nnz, B_nrows,
                                                dB_csrOffsets, HIPSPARSE_INDEX_BASE_ZERO))
                CHECK_CUSPARSE(hipsparseCreateCsr(&matB, B_nrows, B_ncols, B_nnz,
                                                 dB_csrOffsets, dB_columns, dB_values,
                                                 HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                                 HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F))
                CHECK_CUSPARSE(hipsparseSpMM_bufferSize(
                        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &alpha, matB, matZ1, &beta, matZ,
                        HIP_R_64F, HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSizeB))
                CHECK_CUDA(hipMalloc(&dBufferB, bufferSizeB))
                CHECK_CUSPARSE(hipsparseSpMM_preprocess(
                        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &alpha, matB, matZ1, &beta, matZ, HIP_R_64F,
                        HIPSPARSE_SPMM_ALG_DEFAULT, dBufferB))

                for (uint i = 0; i < nobjs; ++i) {
                    thrust::copy(thrust::device, dMasking_nrows, dMasking_nrows + A_nrows, dMasking_tiled + i * A_nrows);
                }
                thrust::copy_if(thrust::device, dR, dR + nobjs * A_nrows,
                                dMasking_tiled, dRPart, mopmc::functions::cuda::is_not_zero<double>());

                iteration = 0;
                do {
                    // Z = R'
                    CHECK_CUBLAS(hipblasDcopy_64(cublasHandle, Z_ncols * Z_nrows, dRPart, 1, dZ, 1))
                    // initialise Z1 as R' too
                    if (iteration == 0) {
                        CHECK_CUBLAS(hipblasDcopy_64(cublasHandle, Z_ncols * Z_nrows, dRPart, 1, dZ1, 1))
                    }
                    // Z = B.Z1 + Z, where Z = R'
                    CHECK_CUSPARSE(hipsparseSpMM(
                            handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            &alpha, matB, matZ1, &beta, matZ,
                            HIP_R_64F, HIPSPARSE_SPMM_ALG_DEFAULT, dBufferB))

                    // Z1 = -1 * Z + Z1
                    CHECK_CUBLAS(hipblasDaxpy_64(cublasHandle, Z_ncols * Z_nrows, &alpha2, dZ, 1, dZ1, 1))
                    // max |Z1|
                    CHECK_CUBLAS(hipblasIdamax(cublasHandle, Z_ncols * Z_nrows, dZ1, 1, &maxInd))
                    // to get maxEps, we must reduce also by one since this is FORTRAN based indexing.
                    CHECK_CUDA(hipMemcpy(&maxEps, dZ1 + maxInd - 1, sizeof(double), hipMemcpyDeviceToHost))
                    maxEps = (maxEps >= 0) ? maxEps : -maxEps;
                    // Z1 = Z
                    CHECK_CUBLAS(hipblasDcopy_64(cublasHandle, Z_ncols * Z_nrows, dZ, 1, dZ1, 1))
                    //printf("___ VI PHASE TWO, OBJECTIVE %i, ITERATION %i, maxEps %f\n", obj, iteration, maxEps);
                    ++iteration;
                } while (maxEps > tolerance && iteration < maxIter);
                if (iteration == maxIter) {
                    std::cout << "[warning] loop exit after reaching maximum iteration number (" << iteration <<")\n";
                }
                // copy results
                for (int obj = 0; obj < nobjs; ++obj) {
                    thrust::copy(thrust::device, dZ + iniRow + obj * Z_nrows, dZ + iniRow + 1 + obj * Z_nrows, dResult + obj);
                }
                //-------------------------------------------------------------------------
                CHECK_CUDA(hipMemcpy(scheduler.data(), dScheduler, A_ncols * sizeof(int), hipMemcpyDeviceToHost))
                CHECK_CUDA(hipMemcpy(results.data(), dResult, (nobjs + 1) * sizeof(double), hipMemcpyDeviceToHost))
                CHECK_CUSPARSE(hipsparseDestroySpMat(matB))
                CHECK_CUDA(hipFree(dBufferB))
                return EXIT_SUCCESS;
            }

            template<typename ValueType>
            int CudaValueIterationHandler<ValueType>::exit() {
                // destroy matrix/vector descriptors
                CHECK_CUSPARSE(hipsparseDestroySpMat(matA))
                CHECK_CUSPARSE(hipsparseDestroyDnVec(vecX))
                CHECK_CUSPARSE(hipsparseDestroyDnVec(vecY))
                CHECK_CUSPARSE(hipsparseDestroyDnVec(vecX1))
                CHECK_CUSPARSE(hipsparseDestroyDnVec(vecRw))
                //CHECK_CUSPARSE(hipsparseDestroySpMat(matB))
                CHECK_CUSPARSE(hipsparseDestroyDnMat(matZ))
                CHECK_CUSPARSE(hipsparseDestroyDnMat(matZ1))
                CHECK_CUSPARSE(hipsparseDestroy(handle))
                // device memory de-allocation
                CHECK_CUDA(hipFree(dBuffer))
                CHECK_CUDA(hipFree(dA_csrOffsets))
                CHECK_CUDA(hipFree(dA_columns))
                CHECK_CUDA(hipFree(dA_values))
                CHECK_CUDA(hipFree(dA_rows_backup))
                CHECK_CUDA(hipFree(dB_csrOffsets))
                CHECK_CUDA(hipFree(dB_columns))
                CHECK_CUDA(hipFree(dB_values))
                CHECK_CUDA(hipFree(dB_rows_backup))
                CHECK_CUDA(hipFree(dX))
                CHECK_CUDA(hipFree(dX1))
                CHECK_CUDA(hipFree(dY))
                CHECK_CUDA(hipFree(dZ))
                CHECK_CUDA(hipFree(dZ1))
                CHECK_CUDA(hipFree(dR))
                CHECK_CUDA(hipFree(dRw))
                CHECK_CUDA(hipFree(dRi))
                CHECK_CUDA(hipFree(dW))
                CHECK_CUDA(hipFree(dRowGroupIndices))
                CHECK_CUDA(hipFree(dRow2RowGroupMapping))
                CHECK_CUDA(hipFree(dMasking_nrows))
                CHECK_CUDA(hipFree(dMasking_nnz))
                CHECK_CUDA(hipFree(dResult))

                std::cout << ("____ CUDA EXIT ____\n");
                return EXIT_SUCCESS;
            }

            /*
            template<typename ValueType>
            __attribute__((unused)) int CudaValueIterationHandler<ValueType>::valueIterationPhaseTwo_deprecated() {
                std::cout << "____ VI PHASE TWO (deprecated) ____\n";
                // generate a DTMC transition matrix as a csr matrix
                CHECK_CUSPARSE(hipsparseXcsr2coo(handle, dA_csrOffsets, A_nnz, A_nrows, dA_rows_backup,
                                                HIPSPARSE_INDEX_BASE_ZERO))

                mopmc::functions::cuda::binaryMaskingLauncher(dA_csrOffsets,
                                                              dRowGroupIndices, dRow2RowGroupMapping,
                                                              dScheduler, dMasking_nrows, dMasking_nnz, A_nrows);
                thrust::copy_if(thrust::device, dA_values, dA_values + A_nnz,
                                dMasking_nnz, dB_values, mopmc::functions::cuda::is_not_zero<int>());
                thrust::copy_if(thrust::device, dA_columns, dA_columns + A_nnz,
                                dMasking_nnz, dB_columns, mopmc::functions::cuda::is_not_zero<int>());
                thrust::copy_if(thrust::device, dA_rows_backup, dA_rows_backup + A_nnz,
                                dMasking_nnz, dB_rows_backup, mopmc::functions::cuda::is_not_zero<int>());
                // @param B_nnz: number of non-zero entries in the DTMC transition matrix
                B_nnz = (int) thrust::count_if(thrust::device, dMasking_nnz, dMasking_nnz + A_nnz,
                                               mopmc::functions::cuda::is_not_zero<double>());
                mopmc::functions::cuda::row2RowGroupLauncher(dRow2RowGroupMapping, dB_rows_backup, B_nnz);
                CHECK_CUSPARSE(hipsparseXcoo2csr(handle, dB_rows_backup, B_nnz, B_nrows,
                                                dB_csrOffsets, HIPSPARSE_INDEX_BASE_ZERO))
                CHECK_CUSPARSE(hipsparseCreateCsr(&matB, B_nrows, B_ncols, B_nnz,
                                                 dB_csrOffsets, dB_columns, dB_values,
                                                 HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                                 HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F))
                // value iteration for all objectives
                // !! As gpu does the main work, we can use mult-threading to send as many
                // individual objective data to gpu as possible.
                for (int obj = 0; obj < nobjs; obj++) {
                    thrust::copy_if(thrust::device, dR + obj * A_nrows, dR + (obj + 1) * A_nrows,
                                    dMasking_nrows, dRi, mopmc::functions::cuda::is_not_zero<double>());
                    iteration = 0;
                    do {
                        // x = ri
                        CHECK_CUBLAS(hipblasDcopy_64(cublasHandle, B_nrows, dRi, 1, dX, 1))
                        // initialise x' as ri too
                        if (iteration == 0) {
                            CHECK_CUBLAS(hipblasDcopy_64(cublasHandle, B_nrows, dRi, 1, dX1, 1))
                        }
                        // x = B.x' + ri where x = ri
                        CHECK_CUSPARSE(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                    &alpha, matB, vecX1, &beta, vecX, HIP_R_64F,
                                                    HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer))
                        // x' = -1 * x + x'
                        CHECK_CUBLAS(hipblasDaxpy_64(cublasHandle, B_ncols, &alpha2, dX, 1, dX1, 1))
                        // max |x'|
                        CHECK_CUBLAS(hipblasIdamax(cublasHandle, A_ncols, dX1, 1, &maxInd))
                        // to get maxEps, we must reduce also by one since this is FORTRAN based indexing.
                        CHECK_CUDA(hipMemcpy(&maxEps, dX1 + maxInd - 1, sizeof(double), hipMemcpyDeviceToHost))
                        maxEps = (maxEps >= 0) ? maxEps : -maxEps;
                        // x' = x
                        CHECK_CUBLAS(hipblasDcopy_64(cublasHandle, B_nrows, dX, 1, dX1, 1))

                        //printf("___ VI PHASE TWO, OBJECTIVE %i, ITERATION %i, maxEps %f\n", obj, iteration, maxEps);
                        ++iteration;

                    } while (maxEps > tolerance && iteration < maxIter);
                    if (iteration == maxIter) {
                        std::cout << "[warning] loop exit after reaching maximum iteration number (" << iteration <<")\n";
                    }
                    //std::cout << "objective " << obj  << " terminated after " << iteration << " iterations\n";
                    // copy results
                    thrust::copy(thrust::device, dX + iniRow, dX + iniRow + 1, dResult + obj);
                }
                //-------------------------------------------------------------------------
                CHECK_CUDA(hipMemcpy(scheduler.data(), dScheduler, A_ncols * sizeof(int), hipMemcpyDeviceToHost))
                CHECK_CUDA(hipMemcpy(results.data(), dResult, (nobjs + 1) * sizeof(double), hipMemcpyDeviceToHost))
                CHECK_CUSPARSE(hipsparseDestroySpMat(matB))
                return EXIT_SUCCESS;
            }
             */

            template class CudaValueIterationHandler<double>;
        }
    }
}