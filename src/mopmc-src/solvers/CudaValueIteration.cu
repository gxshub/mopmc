#include "hip/hip_runtime.h"
//
// Created by guoxin on 15/11/23.
//

#include "CudaValueIteration.cuh"
#include "CuFunctions.h"
//#include <storm/storage/SparseMatrix.h>
//#include <Eigen/Sparse>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>


#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("cuSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUBLAS(func)                                                     \
{                                                                              \
    hipblasStatus_t status = (func);                                            \
    if (status != HIPBLAS_STATUS_SUCCESS) {                                     \
        printf("CUBLAS API failed at line %d with error: %d\n",                \
               __LINE__, status);                                              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

namespace mopmc {
    namespace value_iteration {
        namespace gpu {

            template<typename ValueType>
            CudaValueIterationHandler<ValueType>::CudaValueIterationHandler(
                    const Eigen::SparseMatrix<ValueType, Eigen::RowMajor> &transitionMatrix,
                    const std::vector<int> &rowGroupIndices,
                    const std::vector<int> &row2RowGroupMapping,
                    std::vector<ValueType> &rho_flat,
                    std::vector<int> &pi,
                    int iniRow,
                    int objCount) :
                    transitionMatrix_(transitionMatrix), flattenRewardVector_(rho_flat), scheduler_(pi),
                    rowGroupIndices_(rowGroupIndices), row2RowGroupMapping_(row2RowGroupMapping),
                    iniRow_(iniRow), nobjs(objCount) {

                A_nnz = transitionMatrix_.nonZeros();
                A_ncols = transitionMatrix_.cols();
                A_nrows = transitionMatrix_.rows();
                B_ncols = A_ncols;
                B_nrows = B_ncols;
                C_nrows = B_ncols;
                C_ncols = nobjs;
                C_ld = C_nrows;
                results_.resize(nobjs+1);
                //Assertions
                assert(A_ncols == scheduler_.size());
                assert(flattenRewardVector_.size() == A_nrows * nobjs);
                assert(rowGroupIndices_.size() == A_ncols + 1);
            }

            template<typename ValueType>
            int CudaValueIterationHandler<ValueType>::initialise() {

                // hipMalloc CONSTANTS -------------------------------------------------------------
                CHECK_CUDA(hipMalloc((void **) &dA_csrOffsets, (A_nrows + 1) * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dA_columns, A_nnz * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dA_values, A_nnz * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dA_rows_extra, A_nnz * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dR, A_nrows * nobjs * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dRowGroupIndices, (A_ncols + 1) * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dRow2RowGroupMapping, A_nrows * sizeof(int)))
                // hipMalloc Variables -------------------------------------------------------------
                CHECK_CUDA(hipMalloc((void **) &dX, A_ncols * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dXPrime, A_ncols * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dY, A_nrows * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dPi, A_ncols * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dW, nobjs * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dRw, A_nrows * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dResult, (nobjs + 1) * sizeof(double)))
                // hipMalloc PHASE B-------------------------------------------------------------
                CHECK_CUDA(hipMalloc((void **) &dB_csrOffsets, (A_ncols + 1) * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dB_columns, A_nnz * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dB_values, A_nnz * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dB_rows_extra, A_nnz * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dMasking_nrows, A_nrows * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dMasking_nnz, A_nnz * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dRi, B_nrows * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dRj, nobjs * B_nrows * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dZ, nobjs * A_ncols * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dZPrime, nobjs * A_ncols * sizeof(double)))
                // hipMemcpy -------------------------------------------------------------
                CHECK_CUDA(hipMemcpy(dA_csrOffsets, transitionMatrix_.outerIndexPtr(), (A_nrows + 1) * sizeof(int),
                                      hipMemcpyHostToDevice));
                CHECK_CUDA(hipMemcpy(dA_columns, transitionMatrix_.innerIndexPtr(), A_nnz * sizeof(int),
                                      hipMemcpyHostToDevice));
                CHECK_CUDA(hipMemcpy(dA_values, transitionMatrix_.valuePtr(), A_nnz * sizeof(double),
                                      hipMemcpyHostToDevice));
                CHECK_CUDA(hipMemcpy(dR, flattenRewardVector_.data(), A_nrows * nobjs * sizeof(double),
                                      hipMemcpyHostToDevice));
                CHECK_CUDA(hipMemcpy(dRowGroupIndices, rowGroupIndices_.data(), (A_ncols + 1) * sizeof(int),
                                      hipMemcpyHostToDevice));
                CHECK_CUDA(hipMemcpy(dRow2RowGroupMapping, row2RowGroupMapping_.data(), A_nrows * sizeof(int),
                                      hipMemcpyHostToDevice));
                CHECK_CUDA(hipMemcpy(dPi, scheduler_.data(), A_ncols * sizeof(int), hipMemcpyHostToDevice));
                // NOTE. Data for dW copied in VI phase B.
                //CHECK_CUDA(hipMemset(dY, static_cast<double>(0.0), A_nrows * sizeof(double)))
                //-------------------------------------------------------------------------
                CHECK_CUSPARSE(hipsparseCreate(&handle))
                CHECK_CUBLAS(hipblasCreate(&cublasHandle));
                // Create sparse matrices A in CSR format
                CHECK_CUSPARSE(hipsparseCreateCsr(&matA, A_nrows, A_ncols, A_nnz,
                                                 dA_csrOffsets, dA_columns, dA_values,
                                                 HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                                 HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));
                // Crease dense matrix C
                CHECK_CUSPARSE(hipsparseCreateDnMat(&matC, C_nrows, C_ncols, C_ld, dZ, HIP_R_64F, HIPSPARSE_ORDER_COL));
                // Crease dense matrix D
                CHECK_CUSPARSE(hipsparseCreateDnMat(&matD, C_nrows, C_ncols, C_ld, dZPrime, HIP_R_64F, HIPSPARSE_ORDER_COL));
                // Create dense vector X
                CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, A_ncols, dX, HIP_R_64F))
                // Create dense vector Y
                CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, A_nrows, dY, HIP_R_64F))
                // Create dense vector XPrime
                CHECK_CUSPARSE(hipsparseCreateDnVec(&vecXPrime, A_ncols, dXPrime, HIP_R_64F))
                // Create dense vector Rw
                CHECK_CUSPARSE(hipsparseCreateDnVec(&vecRw, A_nrows, dRw, HIP_R_64F))
                // allocate an external buffer if needed
                CHECK_CUSPARSE(hipsparseSpMV_bufferSize(
                        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                        HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));
                /*
                CHECK_CUSPARSE(hipsparseSpMV_bufferSize(
                        handleB, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &alpha, matB, vecXPrime, &beta, vecX, HIP_R_64F,
                        HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSizeB))
                        */
                CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));
                return EXIT_SUCCESS;
            }

            template<typename ValueType>
            int CudaValueIterationHandler<ValueType>::valueIteration(const std::vector<double> &w) {

                this->valueIterationPhaseOne(w);
                this->valueIterationPhaseTwo();
            }

            template<typename ValueType>
            int CudaValueIterationHandler<ValueType>::valueIterationPhaseOne(const std::vector<double> &w) {

                CHECK_CUDA(hipMemcpy(dW, w.data(), nobjs * sizeof(double), hipMemcpyHostToDevice))
                mopmc::functions::cuda::aggregateLauncher(dW, dR, dRw, A_nrows, nobjs);

                iteration = 0;
                do {
                    // y = r
                    CHECK_CUBLAS(hipblasDcopy_64(cublasHandle, A_nrows, dRw, 1, dY, 1))
                    if (iteration == 0) {
                        mopmc::functions::cuda::maxValueLauncher1(dY, dX, dRowGroupIndices, dPi, A_ncols + 1, A_nrows);
                    }
                    // y = A.x + r (r = y)
                    CHECK_CUSPARSE(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                                                HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer))
                    // x' = x
                    CHECK_CUBLAS(hipblasDcopy_64(cublasHandle, A_ncols, dX, 1, dXPrime, 1))
                    // x(s) = max_{a\in Act(s)} y(s,a), pi(s) = argmax_{a\in Act(s)} pi(s)
                    mopmc::functions::cuda::maxValueLauncher1(dY, dX, dRowGroupIndices, dPi, A_ncols + 1, A_nrows);
                    // x' = -1 * x + x'
                    CHECK_CUBLAS(hipblasDaxpy_64(cublasHandle, A_ncols, &alpha2, dX, 1, dXPrime, 1))
                    // max |x'|
                    CHECK_CUBLAS(hipblasIdamax(cublasHandle, A_ncols, dXPrime, 1, &maxInd))
                    CHECK_CUDA(hipMemcpy(&maxEps, dXPrime + maxInd - 1, sizeof(double), hipMemcpyDeviceToHost))
                    maxEps = (maxEps >= 0) ? maxEps : -maxEps;
                    //maxEps = mopmc::kernels::findMaxEps(dXPrime, A_ncols, maxEps);
                    ++iteration;
                    //printf("___ VI PHASE ONE, ITERATION %i, maxEps %f\n", iteration, maxEps);
                } while (maxEps > 1e-5 && iteration < maxIter);

                printf("___ VI PHASE ONE, terminated at ITERATION %i\n", iteration);
                //copy result
                thrust::copy(thrust::device, dX + iniRow_, dX + iniRow_ + 1, dResult + nobjs);

                return EXIT_SUCCESS;
            }

            template<typename ValueType>
            int CudaValueIterationHandler<ValueType>::valueIterationPhaseTwo() {
                // generate a DTMC transition matrix as a csr matrix
                CHECK_CUSPARSE(hipsparseXcsr2coo(handle, dA_csrOffsets, A_nnz, A_nrows, dA_rows_extra,
                                                HIPSPARSE_INDEX_BASE_ZERO));

                mopmc::functions::cuda::binaryMaskingLauncher(dA_csrOffsets,
                                                              dRowGroupIndices, dRow2RowGroupMapping,
                                                              dPi, dMasking_nrows, dMasking_nnz, A_nrows);
                thrust::copy_if(thrust::device, dA_values, dA_values + A_nnz - 1,
                                dMasking_nnz, dB_values, mopmc::functions::cuda::is_not_zero<int>());
                thrust::copy_if(thrust::device, dA_columns, dA_columns + A_nnz - 1,
                                dMasking_nnz, dB_columns, mopmc::functions::cuda::is_not_zero<int>());
                thrust::copy_if(thrust::device, dA_rows_extra, dA_rows_extra + A_nnz - 1,
                                dMasking_nnz, dB_rows_extra, mopmc::functions::cuda::is_not_zero<int>());
                // @B_nnz: number of non-zero entries in the DTMC transition matrix
                B_nnz = (int) thrust::count_if(thrust::device, dMasking_nnz, dMasking_nnz + A_nnz - 1,
                                               mopmc::functions::cuda::is_not_zero<double>());
                mopmc::functions::cuda::row2RowGroupLauncher(dRow2RowGroupMapping, dB_rows_extra, B_nnz);
                CHECK_CUSPARSE(hipsparseXcoo2csr(handle, dB_rows_extra, B_nnz, B_nrows,
                                                dB_csrOffsets, HIPSPARSE_INDEX_BASE_ZERO));
                CHECK_CUSPARSE(hipsparseCreateCsr(&matB, B_nrows, B_ncols, B_nnz,
                                                 dB_csrOffsets, dB_columns, dB_values,
                                                 HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                                 HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));

                // value iteration for all objectives
                // !! As gpu does the main work, we can use mult-threading to send as many
                // individual objective data to gpu as possible.
                for (int obj = 0; obj < nobjs; obj++) {
                    thrust::copy_if(thrust::device, dR + obj * A_nrows, dR + (obj + 1) * A_nrows - 1,
                                    dMasking_nrows, dRi, mopmc::functions::cuda::is_not_zero<double>());

                    iteration = 0;
                    do {
                        // x = ri
                        CHECK_CUBLAS(hipblasDcopy_64(cublasHandle, B_nrows, dRi, 1, dX, 1));
                        // initialise x' as ri too
                        if (iteration == 0) {
                            CHECK_CUBLAS(hipblasDcopy_64(cublasHandle, B_nrows, dRi, 1, dXPrime, 1));
                        }
                        // x = B.x' + ri where x = ri
                        CHECK_CUSPARSE(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                    &alpha, matB, vecXPrime, &beta, vecX, HIP_R_64F,
                                                    HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer));
                        // x' = -1 * x + x'
                        CHECK_CUBLAS(hipblasDaxpy_64(cublasHandle, B_ncols, &alpha2, dX, 1, dXPrime, 1));
                        // max |x'|
                        CHECK_CUBLAS(hipblasIdamax(cublasHandle, A_ncols, dXPrime, 1, &maxInd));
                        // get maxEps
                        CHECK_CUDA(hipMemcpy(&maxEps, dXPrime + maxInd - 1, sizeof(double), hipMemcpyDeviceToHost));
                        maxEps = (maxEps >= 0) ? maxEps : -maxEps;
                        // x' = x
                        CHECK_CUBLAS(hipblasDcopy_64(cublasHandle, B_nrows, dX, 1, dXPrime, 1));

                        //printf("___ VI PHASE TWO, OBJECTIVE %i, ITERATION %i, maxEps %f\n", obj, iteration, maxEps);
                        ++iteration;

                    } while (maxEps > 1e-5 && iteration < maxIter);
                    printf("___ VI PHASE TWO, OBJECTIVE %i, terminated at ITERATION %i\n", obj, iteration);
                    // copy results
                    thrust::copy(thrust::device, dX + iniRow_, dX + iniRow_ + 1, dResult + obj);
                }

                //-------------------------------------------------------------------------
                CHECK_CUDA(hipMemcpy(scheduler_.data(), dPi, A_ncols * sizeof(int), hipMemcpyDeviceToHost));
                CHECK_CUDA(hipMemcpy(results_.data(), dResult, (nobjs + 1) * sizeof(double), hipMemcpyDeviceToHost));
                return EXIT_SUCCESS;
            }


            template<typename ValueType>
            int CudaValueIterationHandler<ValueType>::exit() {
                // destroy matrix/vector descriptors
                CHECK_CUSPARSE(hipsparseDestroySpMat(matA))
                CHECK_CUSPARSE(hipsparseDestroyDnVec(vecX))
                CHECK_CUSPARSE(hipsparseDestroyDnVec(vecY))
                CHECK_CUSPARSE(hipsparseDestroyDnVec(vecXPrime))
                CHECK_CUSPARSE(hipsparseDestroyDnVec(vecRw))
                CHECK_CUSPARSE(hipsparseDestroySpMat(matB))
                CHECK_CUSPARSE(hipsparseDestroyDnMat(matC))
                CHECK_CUSPARSE(hipsparseDestroyDnMat(matD))
                CHECK_CUSPARSE(hipsparseDestroy(handle))
                // device memory de-allocation
                CHECK_CUDA(hipFree(dBuffer))
                CHECK_CUDA(hipFree(dA_csrOffsets))
                CHECK_CUDA(hipFree(dA_columns))
                CHECK_CUDA(hipFree(dA_values))
                CHECK_CUDA(hipFree(dA_rows_extra))
                CHECK_CUDA(hipFree(dB_csrOffsets))
                CHECK_CUDA(hipFree(dB_columns))
                CHECK_CUDA(hipFree(dB_values))
                CHECK_CUDA(hipFree(dB_rows_extra))
                CHECK_CUDA(hipFree(dX))
                CHECK_CUDA(hipFree(dXPrime))
                CHECK_CUDA(hipFree(dY))
                CHECK_CUDA(hipFree(dZ))
                CHECK_CUDA(hipFree(dZPrime))
                CHECK_CUDA(hipFree(dR))
                CHECK_CUDA(hipFree(dRw))
                CHECK_CUDA(hipFree(dRi))
                CHECK_CUDA(hipFree(dW))
                CHECK_CUDA(hipFree(dRowGroupIndices))
                CHECK_CUDA(hipFree(dRow2RowGroupMapping))
                CHECK_CUDA(hipFree(dMasking_nrows))
                CHECK_CUDA(hipFree(dMasking_nnz))
                CHECK_CUDA(hipFree(dResult))

                printf("____ CUDA EXIT!! ____\n");
                return EXIT_SUCCESS;
            }

            template class CudaValueIterationHandler<double>;
        }
    }
}


/*
{
    int k = 1000;
    std::vector<int> hRowGroupIndices(A_nrows);
    CHECK_CUDA(hipMemcpy(hRowGroupIndices.data(), dRowGroupIndices, k * sizeof(int), hipMemcpyDeviceToHost))
    printf("____ dRowGroupIndices: [");
    for (int i = 0; i < k; ++i) {
        std::cout << hRowGroupIndices[i] << " ";
    }
    std::cout << "...]\n";
}
 */