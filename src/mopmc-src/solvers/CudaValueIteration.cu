#include "hip/hip_runtime.h"
//
// Created by guoxin on 15/11/23.
//

#include "CudaValueIteration.cuh"

#include "CudaOnlyValueIteration.h"
#include "ActionSelection.h"
#include "CuFunctions.h"
#include <storm/storage/SparseMatrix.h>
#include <Eigen/Sparse>
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <thrust/copy.h>
#include <thrust/reduce.h>
#include <thrust/count.h>
#include <thrust/remove.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <iostream>


#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("cuSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUBLAS(func)                                                     \
{                                                                              \
    hipblasStatus_t status = (func);                                            \
    if (status != HIPBLAS_STATUS_SUCCESS) {                                     \
        printf("CUBLAS API failed at line %d with error: %d\n",                \
               __LINE__, status);                                              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

namespace mopmc {
    namespace value_iteration {
        namespace gpu {

            template<typename ValueType>
            CudaValueIterationHandler<ValueType>::CudaValueIterationHandler(
                    const Eigen::SparseMatrix<ValueType, Eigen::RowMajor> &transitionMatrix,
                    std::vector<ValueType> &rho_flat) :
                    transitionMatrix_(transitionMatrix), flattenRewardVector_(rho_flat) {}


            template<typename ValueType>
            CudaValueIterationHandler<ValueType>::CudaValueIterationHandler(
                    const Eigen::SparseMatrix<ValueType, Eigen::RowMajor> &transitionMatrix,
                    const std::vector<int> &rowGroupIndices,
                    const std::vector<int> &row2RowGroupMapping,
                    std::vector<ValueType> &rho_flat,
                    std::vector<int> &pi,
                    std::vector<double> &w,
                    std::vector<double> &x,
                    std::vector<double> &y) :
                    transitionMatrix_(transitionMatrix), flattenRewardVector_(rho_flat), scheduler_(pi),
                    rowGroupIndices_(rowGroupIndices), row2RowGroupMapping_(row2RowGroupMapping),
                    weightVector_(w), x_(x), y_(y) {

                /*
                        {
                    int k = 1000;
                    printf("____ dRow2RowGroupMapping: [");
                    for (int i = 0; i < k; ++i) {
                        std::cout << row2RowGroupMapping_[i] << " ";
                    }
                    std::cout << "...]\n";
                }
                 */
            }


            template<typename ValueType>
            int CudaValueIterationHandler<ValueType>::valueIterationPhaseOne(const std::vector<double> &w) {
                printf("____THIS IS AGGREGATE FUNCTION____\n");
                CHECK_CUDA(hipMemcpy(dW, w.data(), nobjs * sizeof(double), hipMemcpyHostToDevice))
                //CHECK_CUDA(hipMemset(dRw, static_cast<double>(0.0), A_nrows * sizeof(double)))
                mopmc::functions::cuda::aggregateLauncher(dW, dR, dRw, A_nrows, nobjs);

                do {
                    /// y = r
                    CHECK_CUBLAS(hipblasDcopy_64(cublasHandle, A_nrows, dRw, 1, dY, 1))
                    // y = A.x + r (y = r)
                    CHECK_CUSPARSE(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                                                HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer))
                    //
                    // compute the next policy update
                    // x' <- x
                    CHECK_CUBLAS(hipblasDcopy_64(cublasHandle, A_ncols, dX, 1, dXPrime, 1))

                    // x(s) <- max_{a\in Act(s)} y(s,a), pi(s) <- argmax_{a\in Act(s)} pi(s)
                    mopmc::functions::cuda::maxValueLauncher1(dY, dX, dRowGroupIndices, dPi, A_ncols + 1, A_nrows);

                    // x' <- -1 * x + x'
                    CHECK_CUBLAS(hipblasDaxpy_64(cublasHandle, A_ncols, &alpha2, dX, 1, dXPrime, 1))

                    // x(s) <- max_{a\in Act(s)} y(s,a), pi(s) <- argmax_{a\in Act(s)} pi(s)
                    // max |x'|
                    CHECK_CUBLAS(hipblasIdamax(cublasHandle, A_ncols, dXPrime, 1, &maxInd))
                    CHECK_CUDA(hipMemcpy(&maxEps, dXPrime + maxInd - 1, sizeof(double), hipMemcpyDeviceToHost))
                    maxEps = (maxEps >= 0) ? maxEps : -maxEps;
                    //maxEps = mopmc::kernels::findMaxEps(dXPrime, A_ncols, maxEps);
                    //
                    ++iterations;
                    //printf("___ VI PHASE ONE, ITERATION %i, maxEps %f\n", iterations, maxEps);
                } while (maxEps > 1e-5 && iterations < maxIter);

                return EXIT_SUCCESS;
            }

            template<typename ValueType>
            int CudaValueIterationHandler<ValueType>::valueIterationPhaseTwo() {


                // hipMalloc B-------------------------------------------------------------
                CHECK_CUDA(hipMalloc((void **) &dB_csrOffsets, (A_ncols + 1) * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dB_columns, A_nnz * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dB_values, A_nnz * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dB_rows_extra, A_nnz * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dMasking_nrows, A_nrows * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dMasking_nnz, A_nnz * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dRi, B_nrows * sizeof(double)))

                CHECK_CUSPARSE(hipsparseXcsr2coo(handle, dA_csrOffsets, A_nnz, A_nrows, dA_rows_extra,
                                                HIPSPARSE_INDEX_BASE_ZERO));

                mopmc::functions::cuda::maskingLauncher(dA_csrOffsets,
                                                        dRowGroupIndices, dRow2RowGroupMapping,
                                                        dPi, dMasking_nrows, dMasking_nnz, A_nrows);
                /*
                {
                    int k = 1000;
                    std::vector<int> hRowGroupIndices(A_nrows);
                    CHECK_CUDA(hipMemcpy(hRowGroupIndices.data(), dRowGroupIndices, k * sizeof(int), hipMemcpyDeviceToHost))
                    printf("____ dRowGroupIndices: [");
                    for (int i = 0; i < k; ++i) {
                        std::cout << hRowGroupIndices[i] << " ";
                    }
                    std::cout << "...]\n";
                }
                {
                    int k = 1000;
                    std::vector<int> hRow2RowGroupMapping(A_nrows);
                    CHECK_CUDA(hipMemcpy(hRow2RowGroupMapping.data(), dRow2RowGroupMapping, k * sizeof(int), hipMemcpyDeviceToHost))
                    printf("____ dRow2RowGroupMapping: [");
                    for (int i = 0; i < k; ++i) {
                        std::cout << hRow2RowGroupMapping[i] << " ";
                    }
                    std::cout << "...]\n";
                }
                {
                    int k = 1000;
                    std::vector<int> hPi(A_ncols);
                    CHECK_CUDA(hipMemcpy(hPi.data(), dPi, k * sizeof(int), hipMemcpyDeviceToHost))
                    printf("____ dPi: [");
                    for (int i = 0; i < k; ++i) {
                        std::cout << hPi[i] << " ";
                    }
                    std::cout << "...]\n";
                }
                {
                    int k = 1000;
                    std::vector<int> hMasking(A_nnz);
                    CHECK_CUDA(hipMemcpy(hMasking.data(), dMasking_nnz, k * sizeof(int), hipMemcpyDeviceToHost))
                    printf("____ dMasking_nnz: [");
                    for (int i = 0; i < k; ++i) {
                        std::cout << hMasking[i] << " ";
                    }
                    std::cout << "...]\n";
                }
                 */
                {
                    int k = 1000;
                    std::vector<int> hMaskingRows(A_nnz);
                    CHECK_CUDA(hipMemcpy(hMaskingRows.data(), dMasking_nrows, k * sizeof(int), hipMemcpyDeviceToHost))
                    printf("____ dMasking_nrows: [");
                    for (int i = 0; i < k; ++i) {
                        std::cout << hMaskingRows[i] << " ";
                    }
                    std::cout << "...]\n";
                }
                thrust::copy_if(thrust::device, dA_values, dA_values + A_nnz - 1,
                                dMasking_nnz, dB_values, mopmc::functions::cuda::is_not_zero<int>());
                thrust::copy_if(thrust::device, dA_columns, dA_columns + A_nnz - 1,
                                dMasking_nnz, dB_columns, mopmc::functions::cuda::is_not_zero<int>());
                thrust::copy_if(thrust::device, dA_rows_extra, dA_rows_extra + A_nnz - 1,
                                dMasking_nnz, dB_rows_extra, mopmc::functions::cuda::is_not_zero<int>());
                B_nnz = (int) thrust::count_if(thrust::device, dB_values, dB_values + A_nnz - 1,
                                               mopmc::functions::cuda::is_not_zero<double>());
                /*
                {
                    int k = 100;
                    std::vector<int> hB_rows_extra(B_nnz);
                    CHECK_CUDA(hipMemcpy(hB_rows_extra.data(), dB_rows_extra, B_nnz * sizeof(int), hipMemcpyDeviceToHost))
                    printf("____ dB_rows_extra: [");
                    for (int i = 0; i < k; ++i) {
                        std::cout << hB_rows_extra[B_nnz - k + i] << " ";
                    }
                    std::cout << "...]\n";
                }
                 */
                mopmc::functions::cuda::row2RowGroupLauncher(dRow2RowGroupMapping, dB_rows_extra, B_nnz);
                /*
                {
                    int k = 100;
                    std::vector<int> hB_rows_extra(B_nnz);
                    CHECK_CUDA(hipMemcpy(hB_rows_extra.data(), dB_rows_extra, B_nnz * sizeof(int), hipMemcpyDeviceToHost))
                    printf("____ dB_rows_extra: [");
                    for (int i = 0; i < k; ++i) {
                        std::cout << hB_rows_extra[B_nnz - k + i] << " ";
                    }
                    std::cout << "...]\n";
                }
                 */
                std::cout << "A_ncols: " << A_ncols << ", A_nrows: " << A_nrows << ", A_nnz: " << A_nnz << "\n";
                std::cout << "B_ncols: " << B_ncols << ", B_nrows: " << B_nrows << ", B_nnz: " << B_nnz << "\n";

                CHECK_CUSPARSE(hipsparseXcoo2csr(handle, dB_rows_extra,B_nnz, B_nrows,
                                               dB_csrOffsets,HIPSPARSE_INDEX_BASE_ZERO));
                CHECK_CUSPARSE(hipsparseCreateCsr(&matB, B_nrows, B_ncols, B_nnz,
                                                 dB_csrOffsets, dB_columns, dB_values,
                                                 HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                                 HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));
                /*
               CHECK_CUSPARSE(hipsparseCreateCoo(&matB, B_nrows, B_ncols, B_nnz,
                                                dB_rows_extra, dB_columns, dB_values,
                                                HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO,
                                                HIP_R_64F));
               */

                for (int i = 0; i < nobjs; i++) {

                    thrust::copy_if(thrust::device, dR+i*A_nrows, dR+(i+1)*A_nrows-1,
                                    dMasking_nrows, dRi, mopmc::functions::cuda::is_not_zero<double>());

                   /*
                    {
                        int k = 20;
                        std::vector<int> hR(A_nrows);
                        CHECK_CUDA(hipMemcpy(hR.data(), dR, A_nrows * sizeof(double ), hipMemcpyDeviceToHost))
                        printf("____ dR: [... ");
                        for (int i = 0; i < k; ++i) {
                            std::cout << hR[A_nrows - k + i] << " ";
                        }
                        std::cout << "]\n";
                    }

                    {
                        int k = 20;
                        std::vector<int> hRi(B_nrows);
                        CHECK_CUDA(hipMemcpy(hRi.data(), dRi, B_nrows * sizeof(double ), hipMemcpyDeviceToHost))
                        printf("____ dRi: [... ");
                        for (int i = 0; i < k; ++i) {
                            std::cout << hRi[B_nrows - k + i] << " ";
                        }
                        std::cout << "]\n";
                    }
                    */

                    int iterations2 = 0;
                    break;
                    do {
                        /// x' = r
                        CHECK_CUBLAS(hipblasDcopy_64(cublasHandle, B_nrows, dRi, 1, dXPrime, 1));
                        /// x = B.x + r (x' = r)
                        CHECK_CUSPARSE(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                    &alpha, matB, vecXPrime, &beta, vecX, HIP_R_64F,
                                                    HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer));
                        // x' <- -1 * x + x'
                        CHECK_CUBLAS(hipblasDaxpy_64(cublasHandle, B_ncols, &alpha2, dX, 1, dXPrime, 1));
                        // max |x'|
                        CHECK_CUBLAS(hipblasIdamax(cublasHandle, A_ncols, dXPrime, 1, &maxInd));
                        CHECK_CUDA(hipMemcpy(&maxEps, dXPrime + maxInd - 1, sizeof(double), hipMemcpyDeviceToHost));
                        maxEps = (maxEps >= 0) ? maxEps : -maxEps;
                        //printf("___ VI PHASE TWO, ITERATION %i, maxEps %f\n", iterations2, maxEps);
                        ++iterations2;
                    } while (maxEps > 1e-5 && iterations2 < 100);
                }

                return EXIT_SUCCESS;
            }

            template<typename ValueType>
            int CudaValueIterationHandler<ValueType>::initialise() {

                A_nnz = transitionMatrix_.nonZeros();
                A_ncols = transitionMatrix_.cols();
                A_nrows = transitionMatrix_.rows();
                B_ncols = A_ncols;
                B_nrows = B_ncols;
                nobjs = weightVector_.size();
                //Assertions
                assert(A_ncols == x_.size());
                assert(A_ncols == scheduler_.size());
                assert(flattenRewardVector_.size() == A_nrows * nobjs);
                assert(rowGroupIndices_.size() == A_ncols + 1);

                alpha = 1.0;
                beta = 1.0;
                eps = 1.0;
                maxIter = 1000;
                maxEps = 0.0;

                // hipMalloc CONSTANTS -------------------------------------------------------------
                CHECK_CUDA(hipMalloc((void **) &dA_csrOffsets, (A_nrows + 1) * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dA_columns, A_nnz * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dA_values, A_nnz * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dA_rows_extra, A_nnz * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dR, A_nrows * nobjs * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dRowGroupIndices, (A_ncols + 1) * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dRow2RowGroupMapping, A_nrows * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dW, nobjs * sizeof(double)))
                // hipMalloc VARIABLES -------------------------------------------------------------
                CHECK_CUDA(hipMalloc((void **) &dX, A_ncols * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dXPrime, A_ncols * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dY, A_nrows * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dPi, A_ncols * sizeof(int)))
                //CHECK_CUDA(hipMalloc((void **) &dPi_bin, A_nrows * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dRw, A_nrows * sizeof(double)))
                // hipMemcpy -------------------------------------------------------------
                CHECK_CUDA(hipMemcpy(dA_csrOffsets, transitionMatrix_.outerIndexPtr(), (A_nrows + 1) * sizeof(int),
                                      hipMemcpyHostToDevice));
                CHECK_CUDA(hipMemcpy(dA_columns, transitionMatrix_.innerIndexPtr(), A_nnz * sizeof(int),
                                      hipMemcpyHostToDevice));
                CHECK_CUDA(hipMemcpy(dA_values, transitionMatrix_.valuePtr(), A_nnz * sizeof(double),
                                      hipMemcpyHostToDevice))
                CHECK_CUDA(hipMemcpy(dX, x_.data(), A_ncols * sizeof(double), hipMemcpyHostToDevice));
                CHECK_CUDA(hipMemcpy(dXPrime, x_.data(), A_ncols * sizeof(double), hipMemcpyHostToDevice));
                CHECK_CUDA(hipMemcpy(dY, y_.data(), A_nrows * sizeof(double), hipMemcpyHostToDevice));
                //CHECK_CUDA(hipMemset(dY, static_cast<double>(0.0), A_nrows * sizeof(double)))
                CHECK_CUDA(hipMemcpy(dR, flattenRewardVector_.data(), A_nrows * nobjs * sizeof(double),
                                      hipMemcpyHostToDevice));
                CHECK_CUDA(hipMemcpy(dRowGroupIndices, rowGroupIndices_.data(), (A_ncols + 1) * sizeof(int),
                                      hipMemcpyHostToDevice));
                CHECK_CUDA(hipMemcpy(dRow2RowGroupMapping, row2RowGroupMapping_.data(), A_nrows * sizeof(int),
                                      hipMemcpyHostToDevice));
                CHECK_CUDA(hipMemcpy(dPi, scheduler_.data(), A_ncols * sizeof(int), hipMemcpyHostToDevice));
                // NOTE. Data for dW in VI phase 1.
                //-------------------------------------------------------------------------
                CHECK_CUSPARSE(hipsparseCreate(&handle))
                CHECK_CUBLAS(hipblasCreate(&cublasHandle));
                // Create sparse matrices A in CSR format
                CHECK_CUSPARSE(hipsparseCreateCsr(&matA, A_nrows, A_ncols, A_nnz,
                                                 dA_csrOffsets, dA_columns, dA_values,
                                                 HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                                 HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));

                // Create dense vector X
                CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, A_ncols, dX, HIP_R_64F))
                // Create dense vector Y
                CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, A_nrows, dY, HIP_R_64F))
                // Create dense vector XPrime
                CHECK_CUSPARSE(hipsparseCreateDnVec(&vecXPrime, A_ncols, dXPrime, HIP_R_64F))
                // Create dense vector Rw
                CHECK_CUSPARSE(hipsparseCreateDnVec(&vecRw, A_nrows, dRw, HIP_R_64F))
                // allocate an external buffer if needed
                CHECK_CUSPARSE(hipsparseSpMV_bufferSize(
                        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                        HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize));
                /*
                CHECK_CUSPARSE(hipsparseSpMV_bufferSize(
                        handleB, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &alpha, matB, vecXPrime, &beta, vecX, HIP_R_64F,
                        HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSizeB))
                        */
                CHECK_CUDA(hipMalloc(&dBuffer, bufferSize))
                //CHECK_CUDA(hipMalloc(&dBufferB, bufferSizeB))
                //printf("____GOT HERE!!(@) ____\n");
                /////PRINTING
                std::vector<double> dXOut(A_ncols);
                CHECK_CUDA(hipMemcpy(dXOut.data(), dX, A_ncols * sizeof(double), hipMemcpyDeviceToHost))
                return EXIT_SUCCESS;
            }


            template<typename ValueType>
            int CudaValueIterationHandler<ValueType>::exit() {
                CHECK_CUDA(hipMemcpy(scheduler_.data(), dPi, A_ncols * sizeof(int), hipMemcpyDeviceToHost))
                CHECK_CUDA(hipMemcpy(x_.data(), dX, A_ncols * sizeof(double), hipMemcpyDeviceToHost))
                //-------------------------------------------------------------------------
                // destroy matrix/vector descriptors
                CHECK_CUSPARSE(hipsparseDestroySpMat(matA))
                CHECK_CUSPARSE(hipsparseDestroyDnVec(vecX))
                CHECK_CUSPARSE(hipsparseDestroyDnVec(vecY))
                CHECK_CUSPARSE(hipsparseDestroyDnVec(vecXPrime))
                CHECK_CUSPARSE(hipsparseDestroyDnVec(vecRw))
                CHECK_CUSPARSE(hipsparseDestroySpMat(matB))
                CHECK_CUSPARSE(hipsparseDestroy(handle))

                // device memory de-allocation
                CHECK_CUDA(hipFree(dBuffer))
                CHECK_CUDA(hipFree(dA_csrOffsets))
                CHECK_CUDA(hipFree(dA_columns))
                CHECK_CUDA(hipFree(dA_values))
                CHECK_CUDA(hipFree(dA_rows_extra))
                CHECK_CUDA(hipFree(dB_csrOffsets))
                CHECK_CUDA(hipFree(dB_columns))
                CHECK_CUDA(hipFree(dB_values))
                CHECK_CUDA(hipFree(dB_rows_extra))
                CHECK_CUDA(hipFree(dX))
                CHECK_CUDA(hipFree(dXPrime))
                CHECK_CUDA(hipFree(dY))
                CHECK_CUDA(hipFree(dR))
                CHECK_CUDA(hipFree(dRw))
                CHECK_CUDA(hipFree(dRi))
                CHECK_CUDA(hipFree(dW))
                CHECK_CUDA(hipFree(dRowGroupIndices))
                CHECK_CUDA(hipFree(dRow2RowGroupMapping))
                CHECK_CUDA(hipFree(dMasking_nrows))
                CHECK_CUDA(hipFree(dMasking_nnz))


                printf("____ CUDA EXIT!! ____\n");
                return EXIT_SUCCESS;
            }

            template
            class CudaValueIterationHandler<double>;
        }
    }
}
