#include "hip/hip_runtime.h"
//
// Created by guoxin on 15/11/23.
//

#include "CudaValueIteration.cuh"

#include "CudaOnlyValueIteration.h"
#include "ActionSelection.h"
#include "CuFunctions.h"
#include <storm/storage/SparseMatrix.h>
#include <Eigen/Sparse>
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <thrust/copy.h>
#include <thrust/reduce.h>
#include <thrust/count.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <iostream>


#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("cuSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUBLAS(func)                                                     \
{                                                                              \
    hipblasStatus_t status = (func);                                            \
    if (status != HIPBLAS_STATUS_SUCCESS) {                                     \
        printf("CUBLAS API failed at line %d with error: %d\n",                \
               __LINE__, status);                                              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

namespace mopmc { namespace value_iteration { namespace gpu {

    template<typename ValueType>
    CudaValueIterationHandler<ValueType>::CudaValueIterationHandler(const Eigen::SparseMatrix<ValueType, Eigen::RowMajor> &transitionMatrix,
                                                                    std::vector<ValueType> &rho_flat) :
            transitionMatrix_(transitionMatrix), flattenRewardVector(rho_flat) {}


    template<typename ValueType>
    CudaValueIterationHandler<ValueType>::CudaValueIterationHandler(const Eigen::SparseMatrix<ValueType, Eigen::RowMajor> &transitionMatrix,
                                                                    const std::vector<int> &rowGroupIndices,
                                                                    const std::vector<int> &row2RowGroupIndices,
                                                                    std::vector<ValueType> &rho_flat,
                                                                    std::vector<int> &pi,
                                                                    std::vector<double> &w,
                                                                    std::vector<double> &x,
                                                                    std::vector<double> &y) :
            transitionMatrix_(transitionMatrix), flattenRewardVector(rho_flat), scheduler_(pi),
            rowGroupIndices_(rowGroupIndices), row2RowGroupIndices_(row2RowGroupIndices),
            weightVector_(w), x_(x), y_(y) {
    }

    template<typename ValueType>
    int CudaValueIterationHandler<ValueType>::initialise(){

        A_nnz = transitionMatrix_.nonZeros();
        A_ncols = transitionMatrix_.cols();
        A_nrows = transitionMatrix_.rows();
        nobjs = weightVector_.size();
        //Assertions
        assert(A_ncols == x_.size());
        assert(A_ncols == scheduler_.size());
        assert(flattenRewardVector.size() == A_nrows * nobjs);
        assert(rowGroupIndices_.size() == A_ncols + 1);

        alpha = 1.0;
        beta = 1.0;
        eps = 1.0;
        maxIter = 1000;
        maxEps = 0.0;

        // hipMalloc CONSTANTS -------------------------------------------------------------
        CHECK_CUDA(hipMalloc((void **) &dA_csrOffsets, (A_nrows + 1) * sizeof(int)))
        CHECK_CUDA(hipMalloc((void **) &dA_columns, A_nnz * sizeof(int)))
        CHECK_CUDA(hipMalloc((void **) &dA_values, A_nnz * sizeof(double)))
        CHECK_CUDA(hipMalloc((void **) &dA_rows_extra, A_nnz * sizeof(int)))
        CHECK_CUDA(hipMalloc((void **) &dR, A_nrows * nobjs * sizeof(double)))
        CHECK_CUDA(hipMalloc((void **) &dEnabledActions, (A_ncols+1) * sizeof(int)))
        CHECK_CUDA(hipMalloc((void **) &dW, nobjs * sizeof(double)))
        // hipMalloc VARIABLES -------------------------------------------------------------
        CHECK_CUDA(hipMalloc((void **) &dX, A_ncols * sizeof(double)))
        CHECK_CUDA(hipMalloc((void **) &dXPrime, A_ncols * sizeof(double)))
        //CHECK_CUDA(hipMalloc((void **) &dXTemp, A_ncols * sizeof(double))) //TODO dXTemp not needed
        CHECK_CUDA(hipMalloc((void **) &dY, A_nrows * sizeof(double)))
        CHECK_CUDA(hipMalloc((void **) &dPi, A_ncols * sizeof(int)))
        CHECK_CUDA(hipMalloc((void **) &dPi_bin, A_nrows * sizeof(int)))
        CHECK_CUDA(hipMalloc((void **) &dRw, A_nrows * sizeof(double)))
        // hipMalloc B-------------------------------------------------------------
        CHECK_CUDA(hipMalloc((void **) &dB_csrOffsets, (A_ncols + 1) * sizeof(int)))
        CHECK_CUDA(hipMalloc((void **) &dB_columns, A_nnz * sizeof(int)))
        CHECK_CUDA(hipMalloc((void **) &dB_values, A_nnz * sizeof(double)))
        CHECK_CUDA(hipMalloc((void **) &dB_rows_extra, A_nnz * sizeof(int)))
        // hipMemcpy -------------------------------------------------------------
        CHECK_CUDA(hipMemcpy(dA_csrOffsets, transitionMatrix_.outerIndexPtr(),
                              (A_nrows + 1) * sizeof(int), hipMemcpyHostToDevice));
        CHECK_CUDA(hipMemcpy(dA_columns, transitionMatrix_.innerIndexPtr(),
                              A_nnz * sizeof(int), hipMemcpyHostToDevice))
        CHECK_CUDA(hipMemcpy(dA_values, transitionMatrix_.valuePtr(),
                              A_nnz * sizeof(double), hipMemcpyHostToDevice))
        CHECK_CUDA(hipMemcpy(dX, x_.data(), A_ncols * sizeof(double), hipMemcpyHostToDevice))
        CHECK_CUDA(hipMemcpy(dXPrime, x_.data(), A_ncols * sizeof(double), hipMemcpyHostToDevice))
        //CHECK_CUDA(hipMemcpy(dXTemp, x_.data(), A_ncols * sizeof(double), hipMemcpyHostToDevice))
        CHECK_CUDA(hipMemcpy(dY, y_.data(), A_nrows * sizeof(double), hipMemcpyHostToDevice))
        //CHECK_CUDA(hipMemset(dY, static_cast<double>(0.0), A_nrows * sizeof(double)))
        CHECK_CUDA(hipMemcpy(dR, flattenRewardVector.data(), A_nrows * nobjs * sizeof(double), hipMemcpyHostToDevice))
        CHECK_CUDA(hipMemcpy(dEnabledActions, rowGroupIndices_.data(), (A_ncols + 1) * sizeof(int), hipMemcpyHostToDevice))
        CHECK_CUDA(hipMemcpy(dPi, scheduler_.data(), A_ncols * sizeof(int), hipMemcpyHostToDevice))
        // NOTE. Data for dW in VI phase 1.
        //-------------------------------------------------------------------------
        CHECK_CUSPARSE(hipsparseCreate(&handle))
        CHECK_CUBLAS(hipblasCreate(&cublasHandle));
        // Create a sparse matrix A in CSR format
        CHECK_CUSPARSE(hipsparseCreateCsr(&matA, A_nrows, A_ncols, A_nnz,
                                         dA_csrOffsets, dA_columns, dA_values,
                                         HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                         HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F))
        // Create dense vector X
        CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, A_ncols, dX, HIP_R_64F))
        // Create dense vector Y
        CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, A_nrows, dY, HIP_R_64F))
        // Create dense vector Rw
        CHECK_CUSPARSE(hipsparseCreateDnVec(&vecRw, A_nrows, dRw, HIP_R_64F))
        // allocate an external buffer if needed
        CHECK_CUSPARSE(hipsparseSpMV_bufferSize(
                handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize))
        CHECK_CUDA(hipMalloc(&dBuffer, bufferSize))
        //printf("____GOT HERE!!(@) ____\n");
        /////PRINTING
        std::vector<double> dXOut(A_ncols);
        CHECK_CUDA(hipMemcpy(dXOut.data(), dX, A_ncols * sizeof(double), hipMemcpyDeviceToHost))
        return EXIT_SUCCESS;
    }


    template<typename ValueType>
    int CudaValueIterationHandler<ValueType>::exit() {
        CHECK_CUDA(hipMemcpy(scheduler_.data(), dPi, A_ncols * sizeof(int), hipMemcpyDeviceToHost))
        CHECK_CUDA(hipMemcpy(x_.data(), dX, A_ncols * sizeof(double), hipMemcpyDeviceToHost))
        //-------------------------------------------------------------------------
        // destroy matrix/vector descriptors
        CHECK_CUSPARSE(hipsparseDestroySpMat(matA))
        CHECK_CUSPARSE(hipsparseDestroyDnVec(vecX))
        CHECK_CUSPARSE(hipsparseDestroyDnVec(vecY))
        CHECK_CUSPARSE(hipsparseDestroyDnVec(vecRw))
        CHECK_CUSPARSE(hipsparseDestroy(handle))

        // device memory de-allocation
        CHECK_CUDA(hipFree(dBuffer))
        CHECK_CUDA(hipFree(dA_csrOffsets))
        CHECK_CUDA(hipFree(dA_columns))
        CHECK_CUDA(hipFree(dA_values))
        CHECK_CUDA(hipFree(dA_rows_extra))
        CHECK_CUDA(hipFree(dB_csrOffsets))
        CHECK_CUDA(hipFree(dB_columns))
        CHECK_CUDA(hipFree(dB_values))
        CHECK_CUDA(hipFree(dB_rows_extra))
        CHECK_CUDA(hipFree(dX))
        CHECK_CUDA(hipFree(dY))
        CHECK_CUDA(hipFree(dR))
        CHECK_CUDA(hipFree(dRw))
        CHECK_CUDA(hipFree(dW))


        printf("____ CUDA EXIT!! ____\n");
        return EXIT_SUCCESS;
    }

    template<typename ValueType>
    int CudaValueIterationHandler<ValueType>::valueIterationPhaseOne(const std::vector<double> &w){
        printf("____THIS IS AGGREGATE FUNCTION____\n");
        CHECK_CUDA(hipMemcpy(dW, w.data(), nobjs * sizeof(double), hipMemcpyHostToDevice))
        //CHECK_CUDA(hipMemset(dRw, static_cast<double>(0.0), A_nrows * sizeof(double)))
        mopmc::functions::cuda::aggregateLauncher(dW, dR, dRw, A_nrows, nobjs);
        int maxInd = 0;
        int iterations = 0;
        double alpha2 = -1.0;

        do {
            /// y = r
            CHECK_CUBLAS(hipblasDcopy_64(cublasHandle, A_nrows, dRw, 1, dY, 1))
            // y = A.x + r (y = r)
            CHECK_CUSPARSE(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                                        HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer))
            //
            // compute the next policy update
            // x' <- x
            CHECK_CUBLAS(hipblasDcopy_64(cublasHandle, A_ncols, dX, 1, dXPrime, 1))

            // x(s) <- max_{a\in Act(s)} y(s,a), pi(s) <- argmax_{a\in Act(s)} pi(s)
            //mopmc::functions::cuda::maxValueLauncher1(dY, dX, dEnabledActions, dPi, A_ncols+1, A_nrows);
            mopmc::functions::cuda::maxValueLauncher2(dY, dX, dEnabledActions, dPi, dPi_bin, A_ncols + 1);

            // x' <- -1 * x + x'
            CHECK_CUBLAS(hipblasDaxpy_64(cublasHandle, A_ncols, &alpha2, dX, 1, dXPrime, 1))

            // x(s) <- max_{a\in Act(s)} y(s,a), pi(s) <- argmax_{a\in Act(s)} pi(s)
            // max |x'|
            CHECK_CUBLAS(hipblasIdamax(cublasHandle, A_ncols, dXPrime, 1, &maxInd))
            CHECK_CUDA(hipMemcpy(&maxEps, dXPrime+maxInd-1, sizeof(double), hipMemcpyDeviceToHost))
            maxEps = (maxEps >= 0) ? maxEps : -maxEps;
            //maxEps = mopmc::kernels::findMaxEps(dXPrime, A_ncols, maxEps);
            //
            ++iterations;
            printf("___ VI PHASE ONE, ITERATION %i, maxEps %f\n", iterations, maxEps);
        } while (maxEps > 1e-5 && iterations < maxIter);

        return EXIT_SUCCESS;
    }

    template<typename ValueType>
    int CudaValueIterationHandler<ValueType>::valueIterationPhaseTwo() {

        CHECK_CUSPARSE(hipsparseXcsr2coo(handle, dA_csrOffsets, A_nnz, A_nrows, dA_rows_extra, HIPSPARSE_INDEX_BASE_ZERO));

       // thrust::device_ptr< int > dvA_columns = thrust::device_pointer_cast(dA_columns);

        thrust::transform(thrust::device, dA_columns, dA_columns+A_nnz-1,
                          dPi_bin, dB_columns, thrust::multiplies<int>());
        thrust::transform(thrust::device, dA_rows_extra, dA_rows_extra+A_nnz-1,
                          dPi_bin, dB_rows_extra, thrust::multiplies<int>());

        thrust::copy_if(thrust::device, dA_values, dA_values+A_nnz-1,
                        dPi_bin, dB_values, mopmc::functions::cuda::is_not_zero<int>());
        thrust::copy_if(thrust::device, dA_columns, dA_columns+A_nnz-1,
                        dPi_bin, dB_columns, mopmc::functions::cuda::is_not_zero<int>());
        thrust::copy_if(thrust::device, dA_rows_extra, dA_rows_extra+A_nnz-1,
                        dPi_bin, dB_rows_extra, mopmc::functions::cuda::is_not_zero<int>());

        //std::cout << "XXXX: " << thrust::count_if(thrust::device,dA_values,dA_values+A_nnz-1, mopmc::functions::cuda::is_not_zero<double>())<<"\n";

        int B_nnz = (int) thrust::count_if(thrust::device,dB_values,dB_values+A_nnz-1,
                                           mopmc::functions::cuda::is_not_zero<double>());

        /*
        {thrust::device_ptr<double> out_prt = thrust::device_pointer_cast(dB_values);
            for (int i = 0; i < 1000; ++i) {std::cout << "dB_values[" <<i<< "]: " << *(out_prt+i);}
            printf("\n");
        }
         */
        int B_nrows = A_nrows;
        CHECK_CUSPARSE(hipsparseXcoo2csr(handle, dB_rows_extra,B_nnz, B_nrows,
                                        dB_csrOffsets,HIPSPARSE_INDEX_BASE_ZERO));
        for (int i = 0; i < nobjs; i++) {
            int iterations = 0;
            do {


                /*
                // Create a sparse matrix B in CSR format
                CHECK_CUSPARSE(hipsparseCreateCsr(&matB, A_nrows, A_ncols, A_nnz,
                                                 dA_csrOffsets, dA_columns, dA_values,
                                                 HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                                 HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F))
                                                 */

            } while (maxEps > 1e-5 && iterations < maxIter);
        }

        return EXIT_SUCCESS;
    }



    template class CudaValueIterationHandler<double>;
} } }
