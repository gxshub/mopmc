#include "hip/hip_runtime.h"
//
// Created by guoxin on 15/11/23.
//

#include "CudaValueIteration.cuh"
#include "CuFunctions.h"
#include "../solvers/WarmUp.h"
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <thrust/copy.h>
#include <thrust/count.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <iostream>


#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("cuSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUBLAS(func)                                                     \
{                                                                              \
    hipblasStatus_t status = (func);                                            \
    if (status != HIPBLAS_STATUS_SUCCESS) {                                     \
        printf("CUBLAS API failed at line %d with error: %d\n",                \
               __LINE__, status);                                              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

namespace mopmc {
    namespace value_iteration {
        namespace gpu {

            template<typename ValueType>
            CudaValueIterationHandler<ValueType>::CudaValueIterationHandler(QueryData<ValueType, int> *queryData) :
                    data(queryData),
                    transitionMatrix(queryData->transitionMatrix),
                    rowGroupIndices(queryData->rowGroupIndices),
                    row2RowGroupMapping(queryData->row2RowGroupMapping),
                    flattenRewardVector(queryData->flattenRewardVector),
                    scheduler(queryData->scheduler),
                    iniRow(queryData->initialRow),
                    nobjs(queryData->objectiveCount)
            {
                A_nnz = transitionMatrix.nonZeros();
                A_ncols = transitionMatrix.cols();
                A_nrows = transitionMatrix.rows();
                B_ncols = A_ncols;
                B_nrows = B_ncols;
                Z_nrows = B_ncols;
                Z_ncols = nobjs;
                Z_ld = Z_nrows;
                results.resize(nobjs + 1);
                //Assertions
                assert(A_ncols == scheduler.size());
                assert(flattenRewardVector.size() == A_nrows * nobjs);
                assert(rowGroupIndices.size() == A_ncols + 1);
            }

            template<typename ValueType>
            int CudaValueIterationHandler<ValueType>::initialize() {
                //GPU warm up
                mopmc::kernels::launchWarmupKernel();
                std::cout << ("____ CUDA INITIALIZING ____\n");
                // hipMalloc CONSTANTS -------------------------------------------------------------
                CHECK_CUDA(hipMalloc((void **) &dA_csrOffsets, (A_nrows + 1) * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dA_columns, A_nnz * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dA_values, A_nnz * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dA_rows_backup, A_nnz * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dR, A_nrows * nobjs * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dRowGroupIndices, (A_ncols + 1) * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dRow2RowGroupMapping, A_nrows * sizeof(int)))
                // hipMalloc Variables -------------------------------------------------------------
                CHECK_CUDA(hipMalloc((void **) &dX, A_ncols * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dX1, A_ncols * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dY, A_nrows * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dScheduler, A_ncols * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dW, nobjs * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dRw, A_nrows * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dResult, (nobjs + 1) * sizeof(double)))
                // hipMalloc PHASE B-------------------------------------------------------------
                CHECK_CUDA(hipMalloc((void **) &dB_csrOffsets, (A_ncols + 1) * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dB_columns, A_nnz * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dB_values, A_nnz * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dB_rows_backup, A_nnz * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dMasking_nrows, A_nrows * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dMasking_nnz, A_nnz * sizeof(int)))
                CHECK_CUDA(hipMalloc((void **) &dMasking_tiled, Z_ncols * A_nrows * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dRi, B_nrows * sizeof(double))) // this is depreciated, not used in future
                CHECK_CUDA(hipMalloc((void **) &dRPart, Z_ncols * Z_nrows * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dZ, Z_ncols * Z_nrows * sizeof(double)))
                CHECK_CUDA(hipMalloc((void **) &dZ1, Z_ncols * Z_nrows * sizeof(double)))
                // hipMemcpy -------------------------------------------------------------
                CHECK_CUDA(hipMemcpy(dA_csrOffsets, transitionMatrix.outerIndexPtr(), (A_nrows + 1) * sizeof(int),
                                      hipMemcpyHostToDevice))
                CHECK_CUDA(hipMemcpy(dA_columns, transitionMatrix.innerIndexPtr(), A_nnz * sizeof(int),
                                      hipMemcpyHostToDevice))
                CHECK_CUDA(hipMemcpy(dA_values, transitionMatrix.valuePtr(), A_nnz * sizeof(double),
                                      hipMemcpyHostToDevice))
                CHECK_CUDA(hipMemcpy(dR, flattenRewardVector.data(), A_nrows * nobjs * sizeof(double),
                                      hipMemcpyHostToDevice))
                CHECK_CUDA(hipMemcpy(dRowGroupIndices, rowGroupIndices.data(), (A_ncols + 1) * sizeof(int),
                                      hipMemcpyHostToDevice))
                CHECK_CUDA(hipMemcpy(dRow2RowGroupMapping, row2RowGroupMapping.data(), A_nrows * sizeof(int),
                                      hipMemcpyHostToDevice))
                CHECK_CUDA(hipMemcpy(dScheduler, scheduler.data(), A_ncols * sizeof(int), hipMemcpyHostToDevice))
                //-------------------------------------------------------------------------
                CHECK_CUSPARSE(hipsparseCreate(&handle))
                CHECK_CUBLAS(hipblasCreate(&cublasHandle))
                // Create sparse matrices A in CSR format
                CHECK_CUSPARSE(hipsparseCreateCsr(&matA, A_nrows, A_ncols, A_nnz,
                                                 dA_csrOffsets, dA_columns, dA_values,
                                                 HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                                 HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F))
                CHECK_CUSPARSE(hipsparseCreateDnMat(&matZ, Z_nrows, Z_ncols, Z_ld, dZ, HIP_R_64F, HIPSPARSE_ORDER_COL))
                CHECK_CUSPARSE(hipsparseCreateDnMat(&matZ1, Z_nrows, Z_ncols, Z_ld, dZ1, HIP_R_64F, HIPSPARSE_ORDER_COL))
                CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX, A_ncols, dX, HIP_R_64F))
                CHECK_CUSPARSE(hipsparseCreateDnVec(&vecY, A_nrows, dY, HIP_R_64F))
                CHECK_CUSPARSE(hipsparseCreateDnVec(&vecX1, A_ncols, dX1, HIP_R_64F))
                CHECK_CUSPARSE(hipsparseCreateDnVec(&vecRw, A_nrows, dRw, HIP_R_64F))
                // allocate an external buffer if needed
                CHECK_CUSPARSE(hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                       &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                                                       HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize))
                CHECK_CUDA(hipMalloc(&dBuffer, bufferSize))

                return EXIT_SUCCESS;
            }

            template<typename ValueType>
            int CudaValueIterationHandler<ValueType>::valueIteration(const std::vector<double> &w) {

                this->valueIterationPhaseOne(w);
                //this->valueIterationPhaseTwo_deprecated();
                this->valueIterationPhaseTwo();
            }

            template<typename ValueType>
            int CudaValueIterationHandler<ValueType>::valueIterationPhaseOne(const std::vector<double> &w, bool toHost) {
                std::cout << "____ VI PHASE ONE ____\n" ;
                CHECK_CUDA(hipMemcpy(dW, w.data(), nobjs * sizeof(double), hipMemcpyHostToDevice))
                mopmc::functions::cuda::aggregateLauncher(dW, dR, dRw, A_nrows, nobjs);

                iteration = 0;
                do {
                    // Y = R
                    CHECK_CUBLAS(hipblasDcopy_64(cublasHandle, A_nrows, dRw, 1, dY, 1))
                    if (iteration == 0) {
                        mopmc::functions::cuda::maxValueLauncher1(dY, dX, dRowGroupIndices, dScheduler, A_ncols + 1, A_nrows);
                    }
                    // Y = A.X + Y (Y = R)
                    CHECK_CUSPARSE(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                                                HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer))
                    // X1 = X
                    CHECK_CUBLAS(hipblasDcopy_64(cublasHandle, A_ncols, dX, 1, dX1, 1))
                    // X(s) = max_{a\in Act(s)} Y(s,a), scheduler(s) = argmax_{a\in Act(s)} scheduler(s)
                    mopmc::functions::cuda::maxValueLauncher1(dY, dX, dRowGroupIndices, dScheduler, A_ncols + 1, A_nrows);
                    // X1 = -1 * X+ X1
                    CHECK_CUBLAS(hipblasDaxpy_64(cublasHandle, A_ncols, &alpha2, dX, 1, dX1, 1))
                    // max |X1|
                    CHECK_CUBLAS(hipblasIdamax(cublasHandle, A_ncols, dX1, 1, &maxInd))
                    // to get maxEps, we must reduce also by one since this is FORTRAN based indexing.
                    CHECK_CUDA(hipMemcpy(&maxEps, dX1 + maxInd - 1, sizeof(double), hipMemcpyDeviceToHost))
                    maxEps = (maxEps >= 0) ? maxEps : -maxEps;
                    //maxEps = mopmc::kernels::findMaxEps(dXPrime, A_ncols, maxEps);
                    ++iteration;
                    //printf("___ VI PHASE ONE, ITERATION %i, maxEps %f\n", iteration, maxEps);
                } while (maxEps > tolerance && iteration < maxIter);

                if (iteration == maxIter) {
                    std::cout << "[warning] loop exit after reaching maximum iteration number (" << iteration <<")\n";
                }
                //std::cout << "terminated after " << iteration <<" iterations.\n";
                //copy result
                thrust::copy(thrust::device, dX + iniRow, dX + iniRow + 1, dResult + nobjs);
                if(toHost) {
                    CHECK_CUDA(hipMemcpy(scheduler.data(), dScheduler, A_ncols * sizeof(int), hipMemcpyDeviceToHost))
                }

                return EXIT_SUCCESS;
            }

            template<typename ValueType>
            int CudaValueIterationHandler<ValueType>::valueIterationPhaseTwo() {
                std::cout << "____ VI PHASE TWO ____\n";
                // generate a DTMC transition matrix as a csr matrix
                CHECK_CUSPARSE(hipsparseXcsr2coo(handle, dA_csrOffsets, A_nnz, A_nrows, dA_rows_backup,
                                                HIPSPARSE_INDEX_BASE_ZERO))

                mopmc::functions::cuda::binaryMaskingLauncher(dA_csrOffsets,
                                                              dRowGroupIndices, dRow2RowGroupMapping,
                                                              dScheduler, dMasking_nrows, dMasking_nnz, A_nrows);
                thrust::copy_if(thrust::device, dA_values, dA_values + A_nnz,
                                dMasking_nnz, dB_values, mopmc::functions::cuda::is_not_zero<int>());
                thrust::copy_if(thrust::device, dA_columns, dA_columns + A_nnz,
                                dMasking_nnz, dB_columns, mopmc::functions::cuda::is_not_zero<int>());
                thrust::copy_if(thrust::device, dA_rows_backup, dA_rows_backup + A_nnz,
                                dMasking_nnz, dB_rows_backup, mopmc::functions::cuda::is_not_zero<int>());
                /* @param B_nnz: number of non-zero entries in the DTMC transition matrix */
                B_nnz = (int) thrust::count_if(thrust::device, dMasking_nnz, dMasking_nnz + A_nnz,
                                               mopmc::functions::cuda::is_not_zero<double>());
                mopmc::functions::cuda::row2RowGroupLauncher(dRow2RowGroupMapping, dB_rows_backup, B_nnz);
                CHECK_CUSPARSE(hipsparseXcoo2csr(handle, dB_rows_backup, B_nnz, B_nrows,
                                                dB_csrOffsets, HIPSPARSE_INDEX_BASE_ZERO))
                CHECK_CUSPARSE(hipsparseCreateCsr(&matB, B_nrows, B_ncols, B_nnz,
                                                 dB_csrOffsets, dB_columns, dB_values,
                                                 HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                                 HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F))
                CHECK_CUSPARSE(hipsparseSpMM_bufferSize(
                        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &alpha, matB, matZ1, &beta, matZ,
                        HIP_R_64F, HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSizeB))
                CHECK_CUDA(hipMalloc(&dBufferB, bufferSizeB))
                CHECK_CUSPARSE(hipsparseSpMM_preprocess(
                        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                        &alpha, matB, matZ1, &beta, matZ, HIP_R_64F,
                        HIPSPARSE_SPMM_ALG_DEFAULT, dBufferB))

                for (uint i = 0; i < nobjs; ++i) {
                    thrust::copy(thrust::device, dMasking_nrows, dMasking_nrows + A_nrows, dMasking_tiled + i * A_nrows);
                }
                thrust::copy_if(thrust::device, dR, dR + nobjs * A_nrows,
                                dMasking_tiled, dRPart, mopmc::functions::cuda::is_not_zero<double>());

                iteration = 0;
                do {
                    // Z = R'
                    CHECK_CUBLAS(hipblasDcopy_64(cublasHandle, Z_ncols * Z_nrows, dRPart, 1, dZ, 1))
                    // initialise Z1 as R' too
                    if (iteration == 0) {
                        CHECK_CUBLAS(hipblasDcopy_64(cublasHandle, Z_ncols * Z_nrows, dRPart, 1, dZ1, 1))
                    }
                    // Z = B.Z1 + Z, where Z = R'
                    CHECK_CUSPARSE(hipsparseSpMM(
                            handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            &alpha, matB, matZ1, &beta, matZ,
                            HIP_R_64F, HIPSPARSE_SPMM_ALG_DEFAULT, dBufferB))

                    // Z1 = -1 * Z + Z1
                    CHECK_CUBLAS(hipblasDaxpy_64(cublasHandle, Z_ncols * Z_nrows, &alpha2, dZ, 1, dZ1, 1))
                    // max |Z1|
                    CHECK_CUBLAS(hipblasIdamax(cublasHandle, Z_ncols * Z_nrows, dZ1, 1, &maxInd))
                    // to get maxEps, we must reduce also by one since this is FORTRAN based indexing.
                    CHECK_CUDA(hipMemcpy(&maxEps, dZ1 + maxInd - 1, sizeof(double), hipMemcpyDeviceToHost))
                    maxEps = (maxEps >= 0) ? maxEps : -maxEps;
                    // Z1 = Z
                    CHECK_CUBLAS(hipblasDcopy_64(cublasHandle, Z_ncols * Z_nrows, dZ, 1, dZ1, 1))
                    //printf("___ VI PHASE TWO, OBJECTIVE %i, ITERATION %i, maxEps %f\n", obj, iteration, maxEps);
                    ++iteration;

                } while (maxEps > tolerance && iteration < maxIter);
                if (iteration == maxIter) {
                    std::cout << "[warning] loop exit after reaching maximum iteration number (" << iteration <<")\n";
                }

                // copy results
                for (int obj = 0; obj < nobjs; ++obj) {
                    thrust::copy(thrust::device, dZ + iniRow + obj * Z_nrows, dZ + iniRow + 1 + obj * Z_nrows, dResult + obj);
                }
                //-------------------------------------------------------------------------
                CHECK_CUDA(hipMemcpy(scheduler.data(), dScheduler, A_ncols * sizeof(int), hipMemcpyDeviceToHost))
                CHECK_CUDA(hipMemcpy(results.data(), dResult, (nobjs + 1) * sizeof(double), hipMemcpyDeviceToHost))
                CHECK_CUSPARSE(hipsparseDestroySpMat(matB))
                CHECK_CUDA(hipFree(dBufferB))
                return EXIT_SUCCESS;
            }

            template<typename ValueType>
            [[deprecated]] int CudaValueIterationHandler<ValueType>::valueIterationPhaseTwo_deprecated() {
                std::cout << "____ VI PHASE TWO (deprecated) ____\n";
                // generate a DTMC transition matrix as a csr matrix
                CHECK_CUSPARSE(hipsparseXcsr2coo(handle, dA_csrOffsets, A_nnz, A_nrows, dA_rows_backup,
                                                HIPSPARSE_INDEX_BASE_ZERO))

                mopmc::functions::cuda::binaryMaskingLauncher(dA_csrOffsets,
                                                              dRowGroupIndices, dRow2RowGroupMapping,
                                                              dScheduler, dMasking_nrows, dMasking_nnz, A_nrows);
                thrust::copy_if(thrust::device, dA_values, dA_values + A_nnz,
                                dMasking_nnz, dB_values, mopmc::functions::cuda::is_not_zero<int>());
                thrust::copy_if(thrust::device, dA_columns, dA_columns + A_nnz,
                                dMasking_nnz, dB_columns, mopmc::functions::cuda::is_not_zero<int>());
                thrust::copy_if(thrust::device, dA_rows_backup, dA_rows_backup + A_nnz,
                                dMasking_nnz, dB_rows_backup, mopmc::functions::cuda::is_not_zero<int>());
                /* @param B_nnz: number of non-zero entries in the DTMC transition matrix */
                B_nnz = (int) thrust::count_if(thrust::device, dMasking_nnz, dMasking_nnz + A_nnz,
                                               mopmc::functions::cuda::is_not_zero<double>());
                mopmc::functions::cuda::row2RowGroupLauncher(dRow2RowGroupMapping, dB_rows_backup, B_nnz);
                CHECK_CUSPARSE(hipsparseXcoo2csr(handle, dB_rows_backup, B_nnz, B_nrows,
                                                dB_csrOffsets, HIPSPARSE_INDEX_BASE_ZERO))
                CHECK_CUSPARSE(hipsparseCreateCsr(&matB, B_nrows, B_ncols, B_nnz,
                                                 dB_csrOffsets, dB_columns, dB_values,
                                                 HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                                 HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F))

                // value iteration for all objectives
                // !! As gpu does the main work, we can use mult-threading to send as many
                // individual objective data to gpu as possible.
                for (int obj = 0; obj < nobjs; obj++) {
                    thrust::copy_if(thrust::device, dR + obj * A_nrows, dR + (obj + 1) * A_nrows,
                                    dMasking_nrows, dRi, mopmc::functions::cuda::is_not_zero<double>());

                    iteration = 0;
                    do {
                        // x = ri
                        CHECK_CUBLAS(hipblasDcopy_64(cublasHandle, B_nrows, dRi, 1, dX, 1))
                        // initialise x' as ri too
                        if (iteration == 0) {
                            CHECK_CUBLAS(hipblasDcopy_64(cublasHandle, B_nrows, dRi, 1, dX1, 1))
                        }
                        // x = B.x' + ri where x = ri
                        CHECK_CUSPARSE(hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                                    &alpha, matB, vecX1, &beta, vecX, HIP_R_64F,
                                                    HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer))
                        // x' = -1 * x + x'
                        CHECK_CUBLAS(hipblasDaxpy_64(cublasHandle, B_ncols, &alpha2, dX, 1, dX1, 1))
                        // max |x'|
                        CHECK_CUBLAS(hipblasIdamax(cublasHandle, A_ncols, dX1, 1, &maxInd))
                        // to get maxEps, we must reduce also by one since this is FORTRAN based indexing.
                        CHECK_CUDA(hipMemcpy(&maxEps, dX1 + maxInd - 1, sizeof(double), hipMemcpyDeviceToHost))
                        maxEps = (maxEps >= 0) ? maxEps : -maxEps;
                        // x' = x
                        CHECK_CUBLAS(hipblasDcopy_64(cublasHandle, B_nrows, dX, 1, dX1, 1))

                        //printf("___ VI PHASE TWO, OBJECTIVE %i, ITERATION %i, maxEps %f\n", obj, iteration, maxEps);
                        ++iteration;

                    } while (maxEps > tolerance && iteration < maxIter);
                    if (iteration == maxIter) {
                        std::cout << "[warning] loop exit after reaching maximum iteration number (" << iteration <<")\n";
                    }
                    //std::cout << "objective " << obj  << " terminated after " << iteration << " iterations\n";
                    // copy results
                    thrust::copy(thrust::device, dX + iniRow, dX + iniRow + 1, dResult + obj);
                }

                //-------------------------------------------------------------------------
                CHECK_CUDA(hipMemcpy(scheduler.data(), dScheduler, A_ncols * sizeof(int), hipMemcpyDeviceToHost))
                CHECK_CUDA(hipMemcpy(results.data(), dResult, (nobjs + 1) * sizeof(double), hipMemcpyDeviceToHost))
                CHECK_CUSPARSE(hipsparseDestroySpMat(matB))
                return EXIT_SUCCESS;
            }

            /* GS: I commented this out after adopting the idea.
             * But I don't include beginObj and endObj.
             * The two parameters make more sense if we implement a hybrid computing manager.
            template<typename ValueType>
            int CudaValueIterationHandler<ValueType>::valueIterationPhaseTwo_v2(int beginObj, int endObj) {
                // generate a DTMC transition matrix as a csr matrix
                CHECK_CUSPARSE(hipsparseXcsr2coo(handle, dA_csrOffsets, A_nnz, A_nrows, dA_rows_backup,
                                                HIPSPARSE_INDEX_BASE_ZERO));

                mopmc::functions::cuda::binaryMaskingLauncher(dA_csrOffsets,
                                                              dRowGroupIndices, dRow2RowGroupMapping,
                                                              dP, dMasking_nrows, dMasking_nnz, A_nrows);
                thrust::copy_if(thrust::device, dA_values, dA_values + A_nnz,
                                dMasking_nnz, dB_values, mopmc::functions::cuda::is_not_zero<int>());
                thrust::copy_if(thrust::device, dA_columns, dA_columns + A_nnz,
                                dMasking_nnz, dB_columns, mopmc::functions::cuda::is_not_zero<int>());
                thrust::copy_if(thrust::device, dA_rows_backup, dA_rows_backup + A_nnz,
                                dMasking_nnz, dB_rows_backup, mopmc::functions::cuda::is_not_zero<int>());
                // @param B_nnz: number of non-zero entries in the DTMC transition matrix
                B_nnz = (int) thrust::count_if(thrust::device, dMasking_nnz, dMasking_nnz + A_nnz,
                                               mopmc::functions::cuda::is_not_zero<double>());
                mopmc::functions::cuda::row2RowGroupLauncher(dRow2RowGroupMapping, dB_rows_backup, B_nnz);
                CHECK_CUSPARSE(hipsparseXcoo2csr(handle, dB_rows_backup, B_nnz, B_nrows,
                                                dB_csrOffsets, HIPSPARSE_INDEX_BASE_ZERO));
                CHECK_CUSPARSE(hipsparseCreateCsr(&matB, B_nrows, B_ncols, B_nnz,
                                                 dB_csrOffsets, dB_columns, dB_values,
                                                 HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                                 HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));

                // value iteration for all objectives
                // !! As gpu does the main work, we can use mult-threading to send as many
                // individual objective data to gpu as possible.
                //
                // TR: We should avoid this loop and group together windows of the R vector into a matrix
                // one way to do this is just by copying a portion of dR
                //
                double* dRPortion, *dMaskTiled;
                size_t bufferSizeMM;
                void *dBufferMM = nullptr;
                CHECK_CUDA(hipMalloc((void**) &dRPortion, (endObj - beginObj) * B_nrows * sizeof(double)))
                // also tile dMasking_nrows (endObj - beginObj) times
                CHECK_CUDA(hipMalloc((void**) &dMaskTiled, (endObj - beginObj) * A_nrows * sizeof(double)))
                for (uint i = 0; i < (endObj - beginObj); ++i) {
                    thrust::copy(thrust::device, dMasking_nrows, dMasking_nrows + A_nrows, dMaskTiled + i * A_nrows);
                }
                // create a mask of dR based on the tiled dMasking rows starting at the objective index
                thrust::copy_if(thrust::device, dR + beginObj * A_nrows, dR + endObj * A_nrows,
                                dMaskTiled, dRPortion, mopmc::functions::cuda::is_not_zero<double>());
                // Make dRPortion into a dense matrix
                // create a new cuBlas handle => this can be preallocated todo
                double *dY2, *dX2; // TODO rename using proper convention -> insert into initialise once working
                CHECK_CUDA(hipMalloc((void**) &dY2, (endObj - beginObj) * B_nrows * sizeof(double)))
                CHECK_CUDA(hipMalloc((void**) &dX2, (endObj - beginObj) * B_nrows * sizeof(double)))
                CHECK_CUDA(hipMemset(dX2, 0, (endObj - beginObj) * B_nrows * sizeof(double)))
                hipsparseDnMatDescr_t matY2, matX2; //descrR
                // Create the dense matrices with the cusparse dense api
                CHECK_CUSPARSE(hipsparseCreateDnMat(&matY2, B_nrows, (endObj - beginObj), B_nrows,
                                                  dY2, HIP_R_64F, HIPSPARSE_ORDER_COL))
                CHECK_CUSPARSE(hipsparseCreateDnMat(&matX2, B_nrows, (endObj - beginObj), B_nrows,
                                                  dX2, HIP_R_64F, HIPSPARSE_ORDER_COL))

                // set the values of the dense matrix dC to vector dRPortion
                CHECK_CUSPARSE(hipsparseDnMatSetValues(matY2, dRPortion))
                // copy the values of dRPortion whihc is an (endObj - beginOb) * A+ncols
                // in this computation we want to use the kernel formulation
                // C = alpha * op(A) . op(B) + beta * C 
                CHECK_CUSPARSE(hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                        HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, matX2, &beta, matY2, 
                        HIP_R_64F, HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSizeMM))

                CHECK_CUDA(hipMalloc(&dBufferMM, bufferSizeMM));

                CHECK_CUSPARSE(hipsparseSpMM_preprocess(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, 
                               &alpha, matA, matX2, &beta, matY2, HIP_R_64F, 
                               HIPSPARSE_SPMM_ALG_DEFAULT, &dBufferMM))

                CHECK_CUSPARSE(hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               &alpha, matA, matX2, &beta, matY2, HIP_R_64F, 
                               HIPSPARSE_SPMM_ALG_DEFAULT, &dBufferMM))
                ///---///
                // clean up
                CHECK_CUSPARSE(hipsparseDestroyDnMat(matX2))
                CHECK_CUSPARSE(hipsparseDestroyDnMat(matY2))
                CHECK_CUDA(hipFree(dRPortion))
                CHECK_CUDA(hipFree(dMaskTiled))
                CHECK_CUDA(hipFree(dBufferMM))
                //-------------------------------------------------------------------------
                CHECK_CUDA(hipMemcpy(scheduler_.data(), dP, A_ncols * sizeof(int), hipMemcpyDeviceToHost));
                CHECK_CUDA(hipMemcpy(results_.data(), dResult, (nobjs + 1) * sizeof(double), hipMemcpyDeviceToHost));
                return EXIT_SUCCESS;
            }
             */


            template<typename ValueType>
            int CudaValueIterationHandler<ValueType>::exit() {

                // destroy matrix/vector descriptors
                CHECK_CUSPARSE(hipsparseDestroySpMat(matA))
                CHECK_CUSPARSE(hipsparseDestroyDnVec(vecX))
                CHECK_CUSPARSE(hipsparseDestroyDnVec(vecY))
                CHECK_CUSPARSE(hipsparseDestroyDnVec(vecX1))
                CHECK_CUSPARSE(hipsparseDestroyDnVec(vecRw))
                //CHECK_CUSPARSE(hipsparseDestroySpMat(matB))
                CHECK_CUSPARSE(hipsparseDestroyDnMat(matZ))
                CHECK_CUSPARSE(hipsparseDestroyDnMat(matZ1))
                CHECK_CUSPARSE(hipsparseDestroy(handle))
                // device memory de-allocation
                CHECK_CUDA(hipFree(dBuffer))
                CHECK_CUDA(hipFree(dA_csrOffsets))
                CHECK_CUDA(hipFree(dA_columns))
                CHECK_CUDA(hipFree(dA_values))
                CHECK_CUDA(hipFree(dA_rows_backup))
                CHECK_CUDA(hipFree(dB_csrOffsets))
                CHECK_CUDA(hipFree(dB_columns))
                CHECK_CUDA(hipFree(dB_values))
                CHECK_CUDA(hipFree(dB_rows_backup))
                CHECK_CUDA(hipFree(dX))
                CHECK_CUDA(hipFree(dX1))
                CHECK_CUDA(hipFree(dY))
                CHECK_CUDA(hipFree(dZ))
                CHECK_CUDA(hipFree(dZ1))
                CHECK_CUDA(hipFree(dR))
                CHECK_CUDA(hipFree(dRw))
                CHECK_CUDA(hipFree(dRi))
                CHECK_CUDA(hipFree(dW))
                CHECK_CUDA(hipFree(dRowGroupIndices))
                CHECK_CUDA(hipFree(dRow2RowGroupMapping))
                CHECK_CUDA(hipFree(dMasking_nrows))
                CHECK_CUDA(hipFree(dMasking_nnz))
                CHECK_CUDA(hipFree(dResult))

                std::cout << ("____ CUDA EXIT ____\n");
                return EXIT_SUCCESS;
            }

            template class CudaValueIterationHandler<double>;
        }
    }
}


/*
{
    int k = 1000;
    std::vector<int> hRowGroupIndices(A_nrows);
    CHECK_CUDA(hipMemcpy(hRowGroupIndices.data(), dRowGroupIndices, k * sizeof(int), hipMemcpyDeviceToHost))
    printf("____ dRowGroupIndices: [");
    for (int i = 0; i < k; ++i) {
        std::cout << hRowGroupIndices[i] << " ";
    }
    std::cout << "...]\n";
}
 */