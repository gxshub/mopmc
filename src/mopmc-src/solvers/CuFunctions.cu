
#include <hip/hip_runtime.h>
//
// Created by guoxin on 8/11/23.
//

namespace mopmc {
    namespace functions {
        namespace cuda {

            __global__ void aggregate(const double *w, const double *x, double *y, int numRows, int numObjs){
                // y = x * w
                uint tid = threadIdx.x + blockIdx.x * blockDim.x;
                if (tid < numRows) {
                    y[tid] = 0;
                    for (int i = 0; i < numObjs; ++i){
                        y[tid] += w[i] * x[i * numRows + tid];
                    }
                }
            }

            int aggregateLauncher(const double *w, const double *x, double *y, int numRows, int numObjs){
                int blockSize, minGridSize, gridSize;
                hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, &aggregate, 0, numRows);
                gridSize = (numRows + blockSize - 1) / blockSize;
                aggregate<<<gridSize, blockSize>>>(w, x, y, numRows, numObjs);
                return 0;
            }

            __global__ void maxValue1(const double* y, double* x, const int* enabledActions,
                                      int* pi, int arrCount, int numRows) {
                // arrCount is the number of states in the model
                uint tid = threadIdx.x + blockIdx.x * blockDim.x;
                if(tid < arrCount) {
                    // do some stuff
                    int actionStart = enabledActions[tid];
                    int actionEnd = enabledActions[tid+1];
                    /*
                    if(tid < arrCount - 1 ) {
                        actionEnd = enabledActions[tid + 1];
                    } else {
                        actionEnd = numRows;
                    }
                     */
                    int maxIndex = pi[tid];
                    double maxValue = y[actionStart + maxIndex];
                    //double maxValue1 = x[tid];
                    for (int action = 0; action < (actionEnd - actionStart); ++action ) {
                        if (y[actionStart + action] > maxValue) {
                            maxIndex = action;
                            maxValue = y[actionStart+action];
                        }
                    }
                    x[tid] = maxValue;
                    pi[tid] = maxIndex;
                }
            }

            int maxValueLauncher1(double *y, double *x, int *enabledActions, int* pi, int arrCount, int numRows){
                int blockSize;
                int minGridSize;
                int gridSize;

                hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, &maxValue1, 0, arrCount);

                gridSize = (arrCount + blockSize - 1) / blockSize;

                maxValue1<<<gridSize, blockSize>>>(y, x, enabledActions, pi, arrCount, numRows);
                return 0;
            }

            __global__ void maxValue2(const double* y, double* x, const int* enabledActions,
                                      int* pi, int* bpi, int arrCount) {
                // arrCount is the number of states in the model
                uint tid = threadIdx.x + blockIdx.x * blockDim.x;
                if(tid < arrCount) {
                    // do some stuff
                    int actionStart = enabledActions[tid];
                    int actionEnd = enabledActions[tid+1];
                    int maxIndex = pi[tid];
                    double maxValue = y[actionStart + maxIndex];
                    //update pi and x
                    for (int action = 0; action < (actionEnd - actionStart); ++action ) {
                        if (y[actionStart + action] > maxValue) {
                            maxIndex = action;
                            maxValue = y[actionStart+action];
                        }
                    }
                    x[tid] = maxValue;
                    pi[tid] = maxIndex;

                    //update binary pi
                    for (int action = 0; action < (actionEnd - actionStart); ++action ) {
                        if (action == maxIndex) {
                            bpi[actionStart + action] = 1;
                        } else {
                            bpi[actionStart + action] = 0;
                        }
                    }
                }
            }

            int maxValueLauncher2(double *y, double *x, int *enabledActions, int* pi, int* bpi, int arrCount){
                int blockSize;
                int minGridSize;
                int gridSize;

                hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, &maxValue2, 0, arrCount);

                gridSize = (arrCount + blockSize - 1) / blockSize;

                maxValue2<<<gridSize, blockSize>>>(y, x, enabledActions, pi, bpi, arrCount);
                return 0;
            }



            __global__ void abs(const double *x, int k) {
                uint tid = threadIdx.x + blockIdx.x * blockDim.x;
                double diff = 0.0;
                if (tid < k) {
                    if (diff < x[tid] ) {
                        diff = x[tid];
                    }
                    if (diff < -1.0 * x[tid])
                        diff = x[tid];
                }
            }

            int absLauncher(const double *x, int k) {
                int blockSize, minGridSize, gridSize;
                hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, &abs, 0, k);
                gridSize = (blockSize - 1) / blockSize;
                abs<<<gridSize, blockSize>>>(x, k);
                return 0;
            }

        }
    }
}