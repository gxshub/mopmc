
#include <hip/hip_runtime.h>
//
// Created by guoxin on 8/11/23.
//

namespace mopmc {
    namespace functions {
        namespace cuda {

            __global__ void aggregate(const double *w, const double *x, double *z, int numRows, int numObjs){
                // z = x * w
                uint tid = threadIdx.x + blockIdx.x * blockDim.x;
                if (tid < numRows) {
                    z[tid] = 0;
                    for (int i = 0; i < numObjs; ++i){
                        z[tid] += w[i] * x[i * numRows + tid];
                    }
                }
            }

            int aggregateLauncher(const double *w, const double *x, double *z, int numRows, int numObjs){
                int blockSize, minGridSize, gridSize;
                hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, &aggregate, 0, numRows);
                gridSize = (numRows + blockSize - 1) / blockSize;
                aggregate<<<gridSize, blockSize>>>(w, x, z, numRows, numObjs);
                return 0;
            }

            __global__ void maxValue1(const double* y, double* x, const int* enabledActions,
                                      int* pi, int arrCount, int numRows) {
                // arrCount is the number of states in the model
                uint tid = threadIdx.x + blockIdx.x * blockDim.x;
                if(tid < arrCount) {
                    // do some stuff
                    int actionStart = enabledActions[tid];
                    int actionEnd;
                    if(tid < arrCount - 1 ) {
                        actionEnd = enabledActions[tid + 1];
                    } else {
                        actionEnd = numRows;
                    }
                    int maxIndex = pi[tid];
                    double maxValue = y[actionStart + maxIndex];
                    //double maxValue1 = x[tid];
                    for (int action = 0; action < (actionEnd - actionStart); ++action ) {
                        if (y[actionStart + action] > maxValue) {
                            maxIndex = action;
                            maxValue = y[actionStart+action];
                        }
                    }
                    x[tid] = maxValue;
                    pi[tid] = maxIndex;
                }
            }

            int maxValueLauncher1(double *y, double *x, int *enabledActions, int* pi, int arrCount, int numRows){
                int blockSize;
                int minGridSize;
                int gridSize;

                hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, &maxValue1, 0, arrCount);

                gridSize = (arrCount + blockSize - 1) / blockSize;

                maxValue1<<<gridSize, blockSize>>>(y, x, enabledActions, pi, arrCount, numRows);
                return 0;
            }

            __global__ void abs(const double *x, int k) {
                uint tid = threadIdx.x + blockIdx.x * blockDim.x;
                double diff = 0.0;
                if (tid < k) {
                    if (diff < x[tid] ) {
                        diff = x[tid];
                    }
                    if (diff < -1.0 * x[tid])
                        diff = x[tid];
                }
            }

            int absLauncher(const double *x, int k) {
                int blockSize, minGridSize, gridSize;
                hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, &abs, 0, k);
                gridSize = (blockSize - 1) / blockSize;
                abs<<<gridSize, blockSize>>>(x, k);
                return 0;
            }

        }
    }
}